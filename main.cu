#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <fstream>
#include <chrono>

#include "main.h"
#include "cuda_macros.h"
#include <objects.h>

hipError_t execute(Computation* data)
{
    unsigned long long size = CUDA_marshal.variationSize * CUDA_marshal.totalVariations;

    //std::chrono::steady_clock::time_point before = std::chrono::steady_clock::now();
    //std::chrono::steady_clock::time_point precompute, incompute, postcompute;

    hipError_t cudaStatus;

    int blocks = (int)ceil((float)CUDA_marshal.totalVariations / KERNEL_TPB);
    int threads = KERNEL_TPB;
    data->threads_per_block = KERNEL_TPB;

    // We create a dummy CUDA Computation with a Marshal to store the pointers in
    Computation* cuda_computation = nullptr;
    numb* cuda_trajectory = nullptr;
    numb* cuda_parameters = nullptr;
    int* cuda_stepIndices = nullptr;
    numb* cuda_maps2 = nullptr;

    CUDA_SET_DEVICE;

    CUDA_MALLOC(&cuda_computation, sizeof(Computation), "hipMalloc computation failed!");
    CUDA_MALLOC(&cuda_trajectory, size * sizeof(numb), "hipMalloc data failed!");
    CUDA_MALLOC(&cuda_parameters, (unsigned long long)CUDA_marshal.totalVariations * CUDA_kernel.PARAM_COUNT * sizeof(numb), "hipMalloc params failed!");
    CUDA_MALLOC(&cuda_stepIndices, (unsigned long long)CUDA_marshal.totalVariations * CUDA_ATTR_COUNT * sizeof(int), "hipMalloc indices failed!");
    if (CUDA_marshal.totalVariations > 1) CUDA_MALLOC(&cuda_maps2, (unsigned long long)CUDA_marshal.totalVariations * CUDA_kernel.MAP_COUNT * sizeof(numb), "hipMalloc maps2 failed!");

    CUDA_MEMCPY(cuda_computation, data, hipMemcpyHostToDevice, sizeof(Computation), "hipMemcpy computation failed!");
    CUDA_MEMCPY(&(cuda_computation->marshal.trajectory), &cuda_trajectory, hipMemcpyHostToDevice, sizeof(numb*), "hipMemcpy trajectory address failed!");
    CUDA_MEMCPY(&(cuda_computation->marshal.parameterVariations), &cuda_parameters, hipMemcpyHostToDevice, sizeof(numb*), "hipMemcpy parameter address failed!");
    CUDA_MEMCPY(&(cuda_computation->marshal.stepIndices), &cuda_stepIndices, hipMemcpyHostToDevice, sizeof(int*), "hipMemcpy indices address failed!");
    if (CUDA_marshal.totalVariations > 1) CUDA_MEMCPY(&(cuda_computation->marshal.maps2), &cuda_maps2, hipMemcpyHostToDevice, sizeof(numb*), "hipMemcpy maps2 address failed!");

    CUDA_MEMCPY(cuda_trajectory, CUDA_marshal.trajectory, hipMemcpyHostToDevice, size * sizeof(numb), "hipMemcpy data failed!");
    CUDA_MEMCPY(cuda_parameters, CUDA_marshal.parameterVariations, hipMemcpyHostToDevice, (unsigned long long)CUDA_marshal.totalVariations * CUDA_kernel.PARAM_COUNT * sizeof(numb), "hipMemcpy params failed!");
    CUDA_MEMCPY(cuda_stepIndices, CUDA_marshal.stepIndices, hipMemcpyHostToDevice, (unsigned long long)CUDA_marshal.totalVariations * CUDA_ATTR_COUNT * sizeof(int), "hipMemcpy indices failed!");
    if (CUDA_marshal.totalVariations > 1) CUDA_MEMCPY(cuda_maps2, CUDA_marshal.maps2, hipMemcpyHostToDevice, (unsigned long long)CUDA_marshal.totalVariations * CUDA_kernel.MAP_COUNT * sizeof(numb), "hipMemcpy maps2 failed!");

    // Kernel execution
    //precompute = std::chrono::steady_clock::now();
    KERNEL_PROG <<< blocks, threads >>> (cuda_computation);

    CUDA_LASTERROR;

    CUDA_SYNCHRONIZE;
    //incompute = std::chrono::steady_clock::now();

    CUDA_MEMCPY(CUDA_marshal.trajectory, cuda_trajectory, hipMemcpyDeviceToHost, size * sizeof(numb), "hipMemcpy back failed!");
    if (CUDA_marshal.totalVariations > 1) CUDA_MEMCPY(CUDA_marshal.maps2, cuda_maps2, hipMemcpyDeviceToHost, (unsigned long long)CUDA_marshal.totalVariations * CUDA_kernel.MAP_COUNT * sizeof(numb), "hipMemcpy maps back failed!");

Error:
    if (cuda_trajectory != nullptr) hipFree(cuda_trajectory);
    if (cuda_parameters != nullptr) hipFree(cuda_parameters);
    if (cuda_stepIndices != nullptr) hipFree(cuda_stepIndices);
    if (cuda_maps2 != nullptr) hipFree(cuda_maps2);
    if (cuda_computation != nullptr) hipFree(cuda_computation);

    //postcompute = std::chrono::steady_clock::now();
    //printf("Precompute time: %Ii ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(precompute - before).count());
    //printf("Incompute time: %Ii ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(incompute - precompute).count());
    //printf("Postcompute time: %Ii ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(postcompute - incompute).count());

    return cudaStatus;
}

int compute(Computation* data)
{
    std::chrono::steady_clock::time_point before = std::chrono::steady_clock::now();
    std::chrono::steady_clock::time_point after;

    // Preparation
    unsigned long int variations = 1; // Parameter/variable variations (ranging steps)

    CUDA_kernel.CopyFrom(&KERNEL);

    for (int i = 0; i < CUDA_kernel.VAR_COUNT; i++)
        if (CUDA_kernel.variables[i].TrueStepCount() > 1)
            variations *= CUDA_kernel.variables[i].stepCount;

    for (int i = 0; i < CUDA_kernel.PARAM_COUNT; i++)
        if (CUDA_kernel.parameters[i].TrueStepCount() > 1)
            variations *= CUDA_kernel.parameters[i].stepCount;

    CUDA_marshal.totalVariations = variations;
    unsigned long long trajectorySize = CUDA_kernel.VAR_COUNT * (CUDA_kernel.steps + 1); // All steps for the current parameter/variable value combination
    CUDA_marshal.variationSize = (int)trajectorySize;
    unsigned long long size = trajectorySize * variations; // Entire data array size

    if (CUDA_marshal.trajectory == nullptr) CUDA_marshal.trajectory = new numb[size];
    if (CUDA_marshal.parameterVariations == nullptr) CUDA_marshal.parameterVariations = new numb[CUDA_kernel.PARAM_COUNT * variations];
    if (CUDA_marshal.stepIndices == nullptr) CUDA_marshal.stepIndices = new int[CUDA_ATTR_COUNT * variations];

    // Vector of attribute steps (indices of values) is now outside the filling function, this way we can use it in several iterations, essential for hi-res computations
    std::vector<int> attributeStepIndices;
    for (int i = 0; i < CUDA_ATTR_COUNT; i++) attributeStepIndices.push_back(0);

    fillAttributeBuffers(data, attributeStepIndices, 0, variations);
    setMapValues(data);

    bool hasFailed = false;

    // Execution

    hipError_t cudaStatus = execute(data);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "execute failed!\n"); hasFailed = true; }

    // Output

    after = std::chrono::steady_clock::now();
    std::chrono::steady_clock::duration elapsed = after - before;
    auto timeElapsed = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed).count();
    data->timeElapsed = (float)timeElapsed;

    CUDA_RESET;

    return hasFailed;
}


void fillAttributeBuffers(Computation* data, std::vector<int>& attributeStepIndices, unsigned long long startVariation, unsigned long long endVariation)
{
    unsigned long long varStride = CUDA_marshal.variationSize;    // Stride between variations in "trajectory"
    unsigned long long paramStride = CUDA_kernel.PARAM_COUNT;     // Stride between variations in "parameterVariations"

    for (unsigned long long i = startVariation; i < endVariation; i++)
    {
        if (data->isFirst) // Meaning no previous trajectory
        {
            // Forming buffer from attribute values
            // Counting step indeces like a number, incrementing by 1 for each variation
            for (int v = 0; v < CUDA_kernel.VAR_COUNT; v++)
            {
                // i * stride - start of the variation, + v for the variable
                // setting a value from the variable's ranged values, knowing the step index is among the first of "attribute step indices"
                CUDA_marshal.trajectory[i * varStride + v] = CUDA_kernel.variables[v].values[attributeStepIndices[v]];
            }
        }
        else
        {
            // Copying from previous trajectory
            for (int v = 0; v < CUDA_kernel.VAR_COUNT; v++)
            {
                // Left side is the first step of the trajectory
                // Right side is the last step of the previous trajectory
                CUDA_marshal.trajectory[i * varStride + v] = data->otherMarshal->trajectory[i * varStride + (CUDA_kernel.steps * CUDA_kernel.VAR_COUNT) + v];
            }
        }

        for (int p = 0; p < CUDA_kernel.PARAM_COUNT; p++)
            CUDA_marshal.parameterVariations[i * paramStride + p] = CUDA_kernel.parameters[p].values[attributeStepIndices[p + CUDA_kernel.VAR_COUNT]];

        for (int j = 0; j < CUDA_ATTR_COUNT; j++)
            CUDA_marshal.stepIndices[i * CUDA_ATTR_COUNT + j] = attributeStepIndices[j];

        // Incrementing the "attribute step indices" total number
        for (int j = CUDA_ATTR_COUNT - 1; j >= 0; j--)
        {
            attributeStepIndices[j]++;

            bool isParam = j >= CUDA_kernel.VAR_COUNT;
            int stepCountOfAttribute = isParam ?
                CUDA_kernel.parameters[j - CUDA_kernel.VAR_COUNT].TrueStepCount() :
                CUDA_kernel.variables[j].TrueStepCount();

            if (attributeStepIndices[j] < stepCountOfAttribute) break;
            attributeStepIndices[j] = 0;
        }
    }
}

void setMapValues(Computation* data)
{
    // Initialize buffer
    if (CUDA_marshal.totalVariations > 1 && CUDA_kernel.MAP_COUNT > 0 && CUDA_marshal.maps2 == nullptr)
        CUDA_marshal.maps2 = new numb[CUDA_marshal.totalVariations * CUDA_kernel.MAP_COUNT];

    if (CUDA_marshal.totalVariations == 1) for (int m = 0; m < CUDA_kernel.MAP_COUNT; m++)
    {
        CUDA_kernel.mapDatas[m].toCompute = false;
        return;
    }

    // Copy previous map values if present
    if (data->isFirst || !CUDA_kernel.continuousMaps)
    {
        memset(CUDA_marshal.maps2, 0, sizeof(numb) * CUDA_marshal.totalVariations * CUDA_kernel.MAP_COUNT);
    }
    else
    {
        memcpy(CUDA_marshal.maps2, data->otherMarshal->maps2, sizeof(numb) * CUDA_marshal.totalVariations * CUDA_kernel.MAP_COUNT);
    }
}