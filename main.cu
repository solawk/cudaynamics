#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <fstream>
#include <chrono>

#include "main.h"
#include "cuda_macros.h"
#include <objects.h>

hipError_t execute(Computation* data)
{
    // Hi-res only requires limited amount of variations
    unsigned long long variations = !data->isHires ? CUDA_marshal.totalVariations : data->variationsInCurrentExecute;
    unsigned long long size = CUDA_marshal.variationSize * variations;
    int totalMapValues = !data->isHires ? CUDA_marshal.totalMapValuesPerVariation : 1; // We always only calculate one map in hi-res, controlled by "toCompute"s

    //std::chrono::steady_clock::time_point before = std::chrono::steady_clock::now();
    //std::chrono::steady_clock::time_point precompute, incompute, postcompute;

    hipError_t cudaStatus;

    int blocks = (int)ceil((float)variations / KERNEL_TPB);
    int threads = KERNEL_TPB;
    data->threads_per_block = KERNEL_TPB;

    // We create a dummy CUDA Computation with a Marshal to store the pointers in
    Computation* cuda_computation = nullptr;
    numb* cuda_trajectory = nullptr;
    numb* cuda_parameters = nullptr;
    int* cuda_stepIndices = nullptr;
    numb* cuda_maps = nullptr;

    CUDA_SET_DEVICE;

    // Allocation memory on the device for the Computation struct and the buffers
    CUDA_MALLOC(&cuda_computation, sizeof(Computation), "hipMalloc computation failed!");
    CUDA_MALLOC(&cuda_trajectory, size * sizeof(numb), "hipMalloc data failed!");
    CUDA_MALLOC(&cuda_parameters, variations * CUDA_kernel.PARAM_COUNT * sizeof(numb), "hipMalloc params failed!");
    CUDA_MALLOC(&cuda_stepIndices, variations * CUDA_ATTR_COUNT * sizeof(int), "hipMalloc indices failed!");
    if (totalMapValues > 0 && variations > 1) CUDA_MALLOC(&cuda_maps, variations * totalMapValues * sizeof(numb), "hipMalloc maps failed!");

    // Copying the Computation struct to the device
    CUDA_MEMCPY(cuda_computation, data, hipMemcpyHostToDevice, sizeof(Computation), "hipMemcpy computation failed!");
    // Copying addresses of the device-side buffers to the device-side Computation
    CUDA_MEMCPY(&(cuda_computation->marshal.trajectory), &cuda_trajectory, hipMemcpyHostToDevice, sizeof(numb*), "hipMemcpy trajectory address failed!");
    CUDA_MEMCPY(&(cuda_computation->marshal.parameterVariations), &cuda_parameters, hipMemcpyHostToDevice, sizeof(numb*), "hipMemcpy parameter address failed!");
    CUDA_MEMCPY(&(cuda_computation->marshal.stepIndices), &cuda_stepIndices, hipMemcpyHostToDevice, sizeof(int*), "hipMemcpy indices address failed!");
    if (totalMapValues > 0 && variations > 1) CUDA_MEMCPY(&(cuda_computation->marshal.maps), &cuda_maps, hipMemcpyHostToDevice, sizeof(numb*), "hipMemcpy maps address failed!");

    // Copying the values in the buffers themselves to the device
    CUDA_MEMCPY(cuda_trajectory, CUDA_marshal.trajectory, hipMemcpyHostToDevice, size * sizeof(numb), "hipMemcpy data failed!");
    CUDA_MEMCPY(cuda_parameters, CUDA_marshal.parameterVariations, hipMemcpyHostToDevice, variations * CUDA_kernel.PARAM_COUNT * sizeof(numb), "hipMemcpy params failed!");
    CUDA_MEMCPY(cuda_stepIndices, CUDA_marshal.stepIndices, hipMemcpyHostToDevice, variations * CUDA_ATTR_COUNT * sizeof(int), "hipMemcpy indices failed!");
    // We don't need to account for multiple maps since we only calculate one at once
    if (variations > 1) CUDA_MEMCPY(cuda_maps, CUDA_marshal.maps + (!data->isHires ? 0 : data->startVariationInCurrentExecute),
        hipMemcpyHostToDevice, variations * totalMapValues * sizeof(numb), "hipMemcpy maps failed!");

    // Kernel execution
    //precompute = std::chrono::steady_clock::now();
    KERNEL_PROG <<< blocks, threads >>> (cuda_computation);
    CUDA_LASTERROR;
    CUDA_SYNCHRONIZE;
    //incompute = std::chrono::steady_clock::now();

    // Copying the trajectories and the maps back to the host
    CUDA_MEMCPY(CUDA_marshal.trajectory, cuda_trajectory, hipMemcpyDeviceToHost, size * sizeof(numb), "hipMemcpy back failed!");
    if (variations > 1) CUDA_MEMCPY(CUDA_marshal.maps + (!data->isHires ? 0 : data->startVariationInCurrentExecute), cuda_maps,
        hipMemcpyDeviceToHost, variations * totalMapValues * sizeof(numb), "hipMemcpy maps back failed!");

Error:
    if (cuda_trajectory != nullptr) hipFree(cuda_trajectory);
    if (cuda_parameters != nullptr) hipFree(cuda_parameters);
    if (cuda_stepIndices != nullptr) hipFree(cuda_stepIndices);
    if (cuda_maps != nullptr) hipFree(cuda_maps);
    if (cuda_computation != nullptr) hipFree(cuda_computation);

    CUDA_RESET;

    //postcompute = std::chrono::steady_clock::now();
    //printf("Precompute time: %Ii ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(precompute - before).count());
    //printf("Incompute time: %Ii ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(incompute - precompute).count());
    //printf("Postcompute time: %Ii ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(postcompute - incompute).count());

    return cudaStatus;
}

int compute(Computation* data)
{
    std::chrono::steady_clock::time_point before = std::chrono::steady_clock::now();
    std::chrono::steady_clock::time_point after;

    // Preparation
    unsigned long long variations = 1; // Parameter/variable variations (ranging steps)

    for (int i = 0; i < CUDA_kernel.VAR_COUNT; i++)
        if (CUDA_kernel.variables[i].TrueStepCount() > 1)
            variations *= CUDA_kernel.variables[i].stepCount;

    for (int i = 0; i < CUDA_kernel.PARAM_COUNT; i++)
        if (CUDA_kernel.parameters[i].TrueStepCount() > 1)
            variations *= CUDA_kernel.parameters[i].stepCount;

    CUDA_marshal.totalVariations = (int)variations;
    unsigned long long variationSize = CUDA_kernel.VAR_COUNT * (CUDA_kernel.steps + 1); // All steps for the current parameter/variable value combination
    CUDA_marshal.variationSize = (int)variationSize;

    unsigned long long variationsInBuffers = !data->isHires ? variations : data->variationsPerParallelization;
    
    if (CUDA_marshal.trajectory == nullptr) CUDA_marshal.trajectory = new numb[variationSize * variationsInBuffers];
    if (CUDA_marshal.parameterVariations == nullptr) CUDA_marshal.parameterVariations = new numb[CUDA_kernel.PARAM_COUNT * variationsInBuffers];
    if (CUDA_marshal.stepIndices == nullptr) CUDA_marshal.stepIndices = new int[CUDA_ATTR_COUNT * variationsInBuffers];

    // Vector of attribute steps (indices of values) is now outside the filling function, this way we can use it in several iterations, essential for hi-res computations
    int* attributeStepIndices = new int[CUDA_ATTR_COUNT];
    for (int i = 0; i < CUDA_ATTR_COUNT; i++) attributeStepIndices[i] = 0;
    setMapValues(data);

    bool hasFailed = false;
    hipError_t cudaStatus;

    // Execution
    if (!data->isHires)
    {
        fillAttributeBuffers(data, attributeStepIndices, 0, variations, false);
        cudaStatus = execute(data);
        if (cudaStatus != hipSuccess) { fprintf(stderr, "execute failed!\n"); hasFailed = true; }
    }
    else
    {
        data->variationsFinished = 0;
        data->bufferNo = 0;
        data->otherMarshal = &(CUDA_marshal); // We trick it into thinking its own trajectory is the previous trajectory when copying the variable values (ouroboros moment)
        for (unsigned long long v = 0; v < variations; v += data->variationsPerParallelization)
        {
            unsigned long long variationsCurrent = min(variations - v, data->variationsPerParallelization);
            data->variationsInCurrentExecute = variationsCurrent;
            data->startVariationInCurrentExecute = v;

            data->isFirst = true;
            for (int b = 0; b < data->buffersPerVariation; b++)
            {
                fillAttributeBuffers(data, attributeStepIndices, v, v + variationsCurrent, !data->isFirst);
                cudaStatus = execute(data);
                if (cudaStatus != hipSuccess) { fprintf(stderr, "execute failed!\n"); hasFailed = true; break; }
                data->isFirst = false;
            }

            data->variationsFinished = v;
        }
    }

    // Output

    after = std::chrono::steady_clock::now();
    std::chrono::steady_clock::duration elapsed = after - before;
    auto timeElapsed = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed).count();
    data->timeElapsed = (float)timeElapsed;

    delete[] attributeStepIndices;

    return hasFailed;
}


void fillAttributeBuffers(Computation* data, int* attributeStepIndices, unsigned long long startVariation, unsigned long long endVariation, bool onlyTrajectory)
{
    unsigned long long varStride = CUDA_marshal.variationSize;    // Stride between variations in "trajectory"
    unsigned long long paramStride = CUDA_kernel.PARAM_COUNT;     // Stride between variations in "parameterVariations"

    for (unsigned long long i = 0; i < endVariation - startVariation; i++)
    {
        if (data->isFirst) // Meaning no previous trajectory
        {
            // Forming buffer from attribute values
            // Counting step indeces like a number, incrementing by 1 for each variation
            for (int v = 0; v < CUDA_kernel.VAR_COUNT; v++)
            {
                // i * stride - start of the variation, + v for the variable
                // setting a value from the variable's ranged values, knowing the step index is among the first of "attribute step indices"
                CUDA_marshal.trajectory[i * varStride + v] = CUDA_kernel.variables[v].values[attributeStepIndices[v]];
            }
        }
        else
        {
            // Copying from previous trajectory
            for (int v = 0; v < CUDA_kernel.VAR_COUNT; v++)
            {
                // Left side is the first step of the trajectory
                // Right side is the last step of the previous trajectory
                CUDA_marshal.trajectory[i * varStride + v] = data->otherMarshal->trajectory[i * varStride + (CUDA_kernel.steps * CUDA_kernel.VAR_COUNT) + v];
            }
        }

        if (!onlyTrajectory)
        {
            for (int p = 0; p < CUDA_kernel.PARAM_COUNT; p++)
                CUDA_marshal.parameterVariations[i * paramStride + p] = CUDA_kernel.parameters[p].values[attributeStepIndices[p + CUDA_kernel.VAR_COUNT]];

            for (int j = 0; j < CUDA_ATTR_COUNT; j++)
                CUDA_marshal.stepIndices[i * CUDA_ATTR_COUNT + j] = attributeStepIndices[j];

            // Incrementing the "attribute step indices" total number
            for (int j = CUDA_ATTR_COUNT - 1; j >= 0; j--)
            {
                attributeStepIndices[j]++;

                bool isParam = j >= CUDA_kernel.VAR_COUNT;
                int stepCountOfAttribute = isParam ?
                    CUDA_kernel.parameters[j - CUDA_kernel.VAR_COUNT].TrueStepCount() :
                    CUDA_kernel.variables[j].TrueStepCount();

                if (attributeStepIndices[j] < stepCountOfAttribute) break;
                attributeStepIndices[j] = 0;
            }
        }
    }
}

void setMapValues(Computation* data)
{
    if (CUDA_marshal.totalVariations == 1)
        for (int m = 0; m < CUDA_kernel.MAP_COUNT; m++)
        {
            CUDA_kernel.mapDatas[m].toCompute = false;
        }

    // Look through all maps and set their offsets depending on which are to be computed and which are not
    int offset = 0; // Offset is counted in maps to be computed, so it's then multiplied by totalVariations on the device
    for (int m = 0; m < CUDA_kernel.MAP_COUNT; m++)
    {
        if (CUDA_kernel.mapDatas[m].toCompute)
        {
            CUDA_kernel.mapDatas[m].offset = offset;
            offset += CUDA_kernel.mapDatas[m].valueCount;
        }
    }
    CUDA_marshal.totalMapValuesPerVariation = offset;

    // Initialize buffer
    if (CUDA_marshal.totalVariations > 1 && CUDA_kernel.MAP_COUNT > 0 && CUDA_marshal.maps == nullptr)
    {
        CUDA_marshal.maps = new numb[CUDA_marshal.totalVariations * CUDA_marshal.totalMapValuesPerVariation];
    }

    // Copy previous map values if present
    if (data->isFirst || CUDA_kernel.mapWeight == 1.0f)
    {
        memset(CUDA_marshal.maps, 0, sizeof(numb) * CUDA_marshal.totalVariations * CUDA_marshal.totalMapValuesPerVariation);
    }
    else
    {
        memcpy(CUDA_marshal.maps, data->otherMarshal->maps, sizeof(numb) * CUDA_marshal.totalVariations * CUDA_marshal.totalMapValuesPerVariation);
    }
}