#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "izhikevich.h"

namespace attributes
{
    enum variables { v, u, I, t };
    enum parameters { a, b, c, d, p0, p1, p2, p3, Imax, Idc, stepsize, method };
    enum methods { ExplicitEuler };
    enum maps { LLE, MAX, MeanInterval, MeanPeak, Period };
}

__global__ void kernelProgram_izhikevich(Computation* data)
{
    int variation = (blockIdx.x * blockDim.x) + threadIdx.x;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int stepStart, variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    numb variables[MAX_ATTRIBUTES];
    numb variablesNext[MAX_ATTRIBUTES];
    numb parameters[MAX_ATTRIBUTES];
    LOAD_ATTRIBUTES(false);

    // Custom area (usually) starts here

    TRANSIENT_SKIP_NEW(finiteDifferenceScheme_izhikevich);

    for (int s = 0; s < CUDA_kernel.steps && !data->isHires; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;
        finiteDifferenceScheme_izhikevich(FDS_ARGUMENTS);
        RECORD_STEP;
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_izhikevich, MO(LLE));
    }

    if (M(MAX).toCompute)
    {
        MAX_Settings max_settings(MS(MAX, 0));
        MAX(data, max_settings, variation, &finiteDifferenceScheme_izhikevich, MO(MAX));
    }

    if (M(Period).toCompute || M(MeanInterval).toCompute || M(MeanPeak).toCompute)
    {
        DBscan_Settings dbscan_settings(MS(Period, 0), MS(MeanInterval, 0), MS(Period, 1), MS(Period, 2), MS(MeanInterval, 1), MS(MeanInterval, 2), MS(MeanInterval, 3), MS(MeanInterval, 4), P(stepsize));
        Period(data, dbscan_settings, variation, &finiteDifferenceScheme_izhikevich, MO(Period), MO(MeanPeak), MO(MeanInterval));
    }
}

__device__ __forceinline__  void finiteDifferenceScheme_izhikevich(numb* currentV, numb* nextV, numb* parameters, Computation* data)
{
    ifMETHOD(P(method), ExplicitEuler)
    {
        Vnext(I) = fmodf(V(t), P(Idc)) < (0.5f * P(Idc)) ? P(Imax) : 0.0f;
        Vnext(t) = V(t) + P(stepsize);
        Vnext(v) = V(v) + P(stepsize) * (P(p0) * V(v) * V(v) + P(p1) * V(v) + P(p2) - V(u) + Vnext(I));
        Vnext(u) = V(u) + P(stepsize) * (P(a) * (P(b) * V(v) - V(u)));

        if (Vnext(v) >= P(p3))
        {
            Vnext(v) = P(c);
            Vnext(u) = Vnext(u) + P(d);
        }
    }
}