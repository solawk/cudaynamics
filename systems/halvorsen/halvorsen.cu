#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "halvorsen.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { alpha, beta, symmetry, method };
    enum methods { ExplicitEuler, ExplicitMidpoint, ExplicitRungeKutta4, VariableSymmetryCD };
    enum maps { LLE, MAX, MeanInterval, MeanPeak, Period };
}

__global__ void kernelProgram_halvorsen(Computation* data)
{
    int variation = (blockIdx.x * blockDim.x) + threadIdx.x;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int stepStart, variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    numb variables[MAX_ATTRIBUTES];
    numb variablesNext[MAX_ATTRIBUTES];
    numb parameters[MAX_ATTRIBUTES];
    LOAD_ATTRIBUTES(false);

    // Custom area (usually) starts here

    TRANSIENT_SKIP_NEW(finiteDifferenceScheme_halvorsen);

    for (int s = 0; s < CUDA_kernel.steps && !data->isHires; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;
        finiteDifferenceScheme_halvorsen(FDS_ARGUMENTS);
        RECORD_STEP;
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_halvorsen, MO(LLE));
    }

    if (M(MAX).toCompute)
    {
        MAX_Settings max_settings(MS(MAX, 0));
        MAX(data, max_settings, variation, &finiteDifferenceScheme_halvorsen, MO(MAX));
    }

    if (M(Period).toCompute || M(MeanInterval).toCompute || M(MeanPeak).toCompute)
    {
        DBscan_Settings dbscan_settings(MS(Period, 0), MS(MeanInterval, 0), MS(Period, 1), MS(Period, 2), MS(MeanInterval, 1), MS(MeanInterval, 2), MS(MeanInterval, 3), MS(MeanInterval, 4),
            H_BRANCH(parameters[CUDA_kernel.PARAM_COUNT - 1], variables[CUDA_kernel.VAR_COUNT - 1]));
        Period(data, dbscan_settings, variation, &finiteDifferenceScheme_halvorsen, MO(Period), MO(MeanPeak), MO(MeanInterval));
    }
}

__device__ __forceinline__ void finiteDifferenceScheme_halvorsen(numb* currentV, numb* nextV, numb* parameters, Computation* data)
{
    ifMETHOD(P(method), ExplicitEuler)
    {
        Vnext(x) = V(x) + H * (-P(alpha) * V(x) - P(beta) * V(y) - P(beta) * V(z) - V(y) * V(y));
        Vnext(y) = V(y) + H * (-P(alpha) * V(y) - P(beta) * V(z) - P(beta) * V(x) - V(z) * V(z));
        Vnext(z) = V(z) + H * (-P(alpha) * V(z) - P(beta) * V(x) - P(beta) * V(y) - V(x) * V(x));
    }


    ifMETHOD(P(method), ExplicitMidpoint)
    {
        numb xmp = V(x) + 0.5 * H * (-P(alpha) * V(x) - P(beta) * V(y) - P(beta) * V(z) - V(y) * V(y));
        numb ymp = V(y) + 0.5 * H * (-P(alpha) * V(y) - P(beta) * V(z) - P(beta) * V(x) - V(z) * V(z));
        numb zmp = V(z) + 0.5 * H * (-P(alpha) * V(z) - P(beta) * V(x) - P(beta) * V(y) - V(x) * V(x));

        Vnext(x) = V(x) + H * (-P(alpha) * xmp - P(beta) * ymp - P(beta) * zmp - ymp * ymp);
        Vnext(y) = V(y) + H * (-P(alpha) * ymp - P(beta) * zmp - P(beta) * xmp - zmp * zmp);
        Vnext(z) = V(z) + H * (-P(alpha) * zmp - P(beta) * xmp - P(beta) * ymp - xmp * xmp);
    }


    ifMETHOD(P(method), ExplicitRungeKutta4)
    {
        numb kx1 = -P(alpha) * V(x) - P(beta) * V(y) - P(beta) * V(z) - V(y) * V(y);
        numb ky1 = -P(alpha) * V(y) - P(beta) * V(z) - P(beta) * V(x) - V(z) * V(z);
        numb kz1 = -P(alpha) * V(z) - P(beta) * V(x) - P(beta) * V(y) - V(x) * V(x);

        numb xmp = V(x) + 0.5 * H * kx1;
        numb ymp = V(y) + 0.5 * H * ky1;
        numb zmp = V(z) + 0.5 * H * kz1;

        numb kx2 = -P(alpha) * xmp - P(beta) * ymp - P(beta) * zmp - ymp * ymp;
        numb ky2 = -P(alpha) * ymp - P(beta) * zmp - P(beta) * xmp - zmp * zmp;
        numb kz2 = -P(alpha) * zmp - P(beta) * xmp - P(beta) * ymp - xmp * xmp;

        xmp = V(x) + 0.5 * H * kx2;
        ymp = V(y) + 0.5 * H * ky2;
        zmp = V(z) + 0.5 * H * kz2;

        numb kx3 = -P(alpha) * xmp - P(beta) * ymp - P(beta) * zmp - ymp * ymp;
        numb ky3 = -P(alpha) * ymp - P(beta) * zmp - P(beta) * xmp - zmp * zmp;
        numb kz3 = -P(alpha) * zmp - P(beta) * xmp - P(beta) * ymp - xmp * xmp;

        xmp = V(x) + H * kx3;
        ymp = V(y) + H * ky3;
        zmp = V(z) + H * kz3;

        numb kx4 = -P(alpha) * xmp - P(beta) * ymp - P(beta) * zmp - ymp * ymp;
        numb ky4 = -P(alpha) * ymp - P(beta) * zmp - P(beta) * xmp - zmp * zmp;
        numb kz4 = -P(alpha) * zmp - P(beta) * xmp - P(beta) * ymp - xmp * xmp;

        Vnext(x) = V(x) + H * (kx1 + 2 * kx2 + 2 * kx3 + kx4) / 6;
        Vnext(y) = V(y) + H * (ky1 + 2 * ky2 + 2 * ky3 + ky4) / 6;
        Vnext(z) = V(z) + H * (kz1 + 2 * kz2 + 2 * kz3 + kz4) / 6;
    }

    ifMETHOD(P(method), VariableSymmetryCD)
    {
        numb h1 = 0.5 * H - P(symmetry);
        numb h2 = 0.5 * H + P(symmetry);

        numb xmp = V(x) + h1 * (-P(alpha) * V(x) - P(beta) * V(y) - P(beta) * V(z) - V(y) * V(y));
        numb ymp = V(y) + h1 * (-P(alpha) * V(y) - P(beta) * V(z) - P(beta) * xmp - V(z) * V(z));
        numb zmp = V(z) + h1 * (-P(alpha) * V(z) - P(beta) * xmp - P(beta) * ymp - xmp * xmp);

        Vnext(z) = (zmp - h2 * (P(beta) * (xmp + ymp) + xmp * xmp)) / (1 + h2 * P(alpha));
        Vnext(y) = (ymp - h2 * (P(beta) * (Vnext(z) + xmp) + Vnext(z) * Vnext(z))) / (1 + h2 * P(alpha));
        Vnext(x) = (xmp - h2 * (P(beta) * (Vnext(y) + Vnext(z)) + Vnext(y) * Vnext(y))) / (1 + h2 * P(alpha));
    }

}