#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "hindmarsh_rose.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { a, b, c, d, r, s, e, Iext, symmetry, method, COUNT };
    enum methods { ExplicitEuler, ExplicitMidpoint, ExplicitRungeKutta4, VariableSymmetryCD};
    enum maps { LLE, MAX, MeanInterval, MeanPeak, Period };
}

__global__ void kernelProgram_hindmarsh_rose(Computation* data)
{
    int variation = (blockIdx.x * blockDim.x) + threadIdx.x;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int stepStart, variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    numb variables[MAX_ATTRIBUTES];
    numb variablesNext[MAX_ATTRIBUTES];
    numb parameters[MAX_ATTRIBUTES];
    LOAD_ATTRIBUTES(false);

    // Custom area (usually) starts here

    TRANSIENT_SKIP_NEW(finiteDifferenceScheme_hindmarsh_rose);

    for (int s = 0; s < CUDA_kernel.steps && !data->isHires; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;
        finiteDifferenceScheme_hindmarsh_rose(FDS_ARGUMENTS);
        RECORD_STEP;
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_hindmarsh_rose, MO(LLE));
    }

    if (M(MAX).toCompute)
    {
        MAX_Settings max_settings(MS(MAX, 0));
        MAX(data, max_settings, variation, &finiteDifferenceScheme_hindmarsh_rose, MO(MAX));
    }

    if (M(Period).toCompute || M(MeanInterval).toCompute || M(MeanPeak).toCompute)
    {
        DBSCAN_Settings dbscan_settings(MS(Period, 0), MS(MeanInterval, 0), MS(Period, 1), MS(Period, 2), MS(MeanInterval, 1), MS(MeanInterval, 2), MS(MeanInterval, 3), MS(MeanInterval, 4),
            H);
        Period(data, dbscan_settings, variation, &finiteDifferenceScheme_hindmarsh_rose, MO(Period), MO(MeanPeak), MO(MeanInterval));
    }
}

__device__ __forceinline__  void finiteDifferenceScheme_hindmarsh_rose(numb* currentV, numb* nextV, numb* parameters)
{
    ifMETHOD(P(method), ExplicitEuler)
    {
        Vnext(x) = V(x) + H * (V(y) - P(a) * V(x) * V(x) * V(x) + P(b) * V(x) * V(x) - V(z) + P(Iext));
        Vnext(y) = V(y) + H * (P(c) - P(d) * V(x) * V(x) - V(y));
        Vnext(z) = V(z) + H * (P(r) * (P(s) * (V(x) + P(e)) - V(z)));
    }

    ifMETHOD(P(method), ExplicitMidpoint)
    {
        numb xmp = V(x) + H * 0.5 * (V(y) - P(a) * V(x) * V(x) * V(x) + P(b) * V(x) * V(x) - V(z) + P(Iext));
        numb ymp = V(y) + H * 0.5 * (P(c) - P(d) * V(x) * V(x) - V(y));
        numb zmp = V(z) + H * 0.5 * (P(r) * (P(s) * (V(x) + P(e)) - V(z)));

        Vnext(x) = V(x) + H * (ymp - P(a) * xmp * xmp * xmp + P(b) * xmp * xmp - zmp + P(Iext));
        Vnext(y) = V(y) + H * (P(c) - P(d) * xmp * xmp - ymp);
        Vnext(z) = V(z) + H * (P(r) * (P(s) * (xmp + P(e)) - zmp));
    }

    ifMETHOD(P(method), ExplicitRungeKutta4)
    {
        numb kx1 = V(y) - P(a) * V(x) * V(x) * V(x) + P(b) * V(x) * V(x) - V(z) + P(Iext);
        numb ky1 = P(c) - P(d) * V(x) * V(x) - V(y);
        numb kz1 = P(r) * (P(s) * (V(x) + P(e)) - V(z));

        numb xmp = V(x) + 0.5 * H * kx1;
        numb ymp = V(y) + 0.5 * H * ky1;
        numb zmp = V(z) + 0.5 * H * kz1;

        numb kx2 = ymp - P(a) * xmp * xmp * xmp + P(b) * xmp * xmp - zmp + P(Iext);
        numb ky2 = P(c) - P(d) * xmp * xmp - ymp;
        numb kz2 = P(r) * (P(s) * (xmp + P(e)) - zmp);

        xmp = V(x) + 0.5 * H * kx2;
        ymp = V(y) + 0.5 * H * ky2;
        zmp = V(z) + 0.5 * H * kz2;

        numb kx3 = ymp - P(a) * xmp * xmp * xmp + P(b) * xmp * xmp - zmp + P(Iext);
        numb ky3 = P(c) - P(d) * xmp * xmp - ymp;
        numb kz3 = P(r) * (P(s) * (xmp + P(e)) - zmp);

        xmp = V(x) + H * kx3;
        ymp = V(y) + H * ky3;
        zmp = V(z) + H * kz3;

        numb kx4 = ymp - P(a) * xmp * xmp * xmp + P(b) * xmp * xmp - zmp + P(Iext);
        numb ky4 = P(c) - P(d) * xmp * xmp - ymp;
        numb kz4 = P(r) * (P(s) * (xmp + P(e)) - zmp);

        Vnext(x) = V(x) + H * (kx1 + 2.0 * kx2 + 2.0 * kx3 + kx4) / 6.0;
        Vnext(y) = V(y) + H * (ky1 + 2.0 * ky2 + 2.0 * ky3 + ky4) / 6.0;
        Vnext(z) = V(z) + H * (kz1 + 2.0 * kz2 + 2.0 * kz3 + kz4) / 6.0;
    }

    ifMETHOD(P(method), VariableSymmetryCD)
    {
        numb h1 = 0.5 * H - P(symmetry);
        numb h2 = 0.5 * H + P(symmetry);

        numb xmp = V(x) + h1 * (V(y) - P(a) * V(x) * V(x) * V(x) + P(b) * V(x) * V(x) - V(z) + P(Iext));
        numb ymp = V(y) + h1 * (P(c) - P(d) * xmp * xmp - V(y));
        numb zmp = V(z) + h1 * (P(r) * (P(s) * (xmp + P(e)) - V(z)));

        Vnext(z) = (zmp + P(r) * P(s) * (xmp + P(e)) * h2) / (1 + P(r) * h2);
        Vnext(y) = (ymp + (P(c) - P(d) * xmp * xmp) * h2) / (1 + h2);

        Vnext(x) = xmp + h2 * (Vnext(y) - P(a) * xmp * xmp * xmp + P(b) * xmp * xmp - Vnext(z) + P(Iext));
        Vnext(x) = xmp + h2 * (Vnext(y) - P(a) * Vnext(x) * Vnext(x) * Vnext(x) + P(b) * Vnext(x) * Vnext(x) - Vnext(z) + P(Iext));
    }
}
