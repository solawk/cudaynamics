#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "rossler.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { a, b, c, stepsize, symmetry, method };
    enum methods { ExplicitEuler, ExplicitMidpoint, ExplicitRungeKutta4, VariableSymmetryCD};
    enum maps { LLE, MAX, MeanInterval, MeanPeak, Period };
}

__global__ void kernelProgram_rossler(Computation* data)
{
    
    int variation = (blockIdx.x * blockDim.x) + threadIdx.x;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int stepStart, variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    numb variables[MAX_ATTRIBUTES];
    numb variablesNext[MAX_ATTRIBUTES];
    numb parameters[MAX_ATTRIBUTES];
    LOAD_ATTRIBUTES;

    // Custom area (usually) starts here

    TRANSIENT_SKIP_NEW(finiteDifferenceScheme_rossler);

    for (int s = 0; s < CUDA_kernel.steps && !data->isHires; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;
        finiteDifferenceScheme_rossler(FDS_ARGUMENTS);
        RECORD_STEP;
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_rossler, MO(LLE));
    }

    if (M(MAX).toCompute)
    {
        MAX_Settings max_settings(MS(MAX, 0));
        MAX(data, max_settings, variation, &finiteDifferenceScheme_rossler, MO(MAX));
    }

    if (M(Period).toCompute || M(MeanInterval).toCompute || M(MeanPeak).toCompute)
    {
        DBscan_Settings dbscan_settings(MS(Period, 0), MS(MeanInterval, 0), MS(Period, 1), MS(Period, 2), MS(MeanInterval, 1), MS(MeanInterval, 2), MS(MeanInterval, 3), MS(MeanInterval, 4), P(stepsize));
        Period(data, dbscan_settings, variation, &finiteDifferenceScheme_rossler,  MO(Period), MO(MeanPeak), MO(MeanInterval));
    }
}

__device__ __forceinline__ void finiteDifferenceScheme_rossler(numb* currentV, numb* nextV, numb* parameters, Computation* data)
{
    ifMETHOD(P(method), ExplicitEuler)
    {
        Vnext(x) = V(x) + P(stepsize) * (-(V(y) + V(z)));
        Vnext(y) = V(y) + P(stepsize) * (V(x) + P(a) * V(y));
        Vnext(z) = V(z) + P(stepsize) * (P(b) + V(z) * (V(x) - P(c)));
    }

    ifMETHOD(P(method), ExplicitMidpoint)
    {
        numb xmp = V(x) + P(stepsize) * 0.5 * (-(V(y) + V(z)));
        numb ymp = V(y) + P(stepsize) * 0.5 * (V(x) + P(a) * V(y));
        numb zmp = V(z) + P(stepsize) * 0.5 * (P(b) + V(z) * (V(x) - P(c)));

        Vnext(x) = V(x) + P(stepsize) * (-(ymp + zmp));
        Vnext(y) = V(y) + P(stepsize) * (xmp + P(a) * ymp);
        Vnext(z) = V(z) + P(stepsize) * (P(b) + zmp * (xmp - P(c)));
    }

    ifMETHOD(P(method), ExplicitRungeKutta4)
    {
        numb kx1 = -(V(y) + V(z));
        numb ky1 = V(x) + P(a) * V(y);
        numb kz1 = P(b) + V(z) * (V(x) - P(c));

        numb xmp = V(x) + 0.5 * P(stepsize) * kx1;
        numb ymp = V(y) + 0.5 * P(stepsize) * ky1;
        numb zmp = V(z) + 0.5 * P(stepsize) * kz1;

        numb kx2 = -(ymp + zmp);
        numb ky2 = xmp + P(a) * ymp;
        numb kz2 = P(b) + zmp * (xmp - P(c));

        xmp = V(x) + 0.5 * P(stepsize) * kx2;
        ymp = V(y) + 0.5 * P(stepsize) * ky2;
        zmp = V(z) + 0.5 * P(stepsize) * kz2;

        numb kx3 = -(ymp + zmp);
        numb ky3 = xmp + P(a) * ymp;
        numb kz3 = P(b) + zmp * (xmp - P(c));

        xmp = V(x) + P(stepsize) * kx3;
        ymp = V(y) + P(stepsize) * ky3;
        zmp = V(z) + P(stepsize) * kz3;

        numb kx4 = -(ymp + zmp);
        numb ky4 = xmp + P(a) * ymp;
        numb kz4 = P(b) + zmp * (xmp - P(c));

        Vnext(x) = V(x) + P(stepsize) * (kx1 + 2 * kx2 + 2 * kx3 + kx4) / 6;
        Vnext(y) = V(y) + P(stepsize) * (ky1 + 2 * ky2 + 2 * ky3 + ky4) / 6;
        Vnext(z) = V(z) + P(stepsize) * (kz1 + 2 * kz2 + 2 * kz3 + kz4) / 6;
    }

    ifMETHOD(P(method), VariableSymmetryCD)
    {
        numb h1 = 0.5 * P(stepsize) - P(symmetry);
        numb h2 = 0.5 * P(stepsize) + P(symmetry);

        numb zmp = V(z) + h1 * (P(b) + V(z) * (V(x) - P(c)));
        numb ymp = V(y) + h1 * (V(x) + P(a) * V(y));
        numb xmp = V(x) + h1 * (-(ymp + zmp));
        
        Vnext(x) = xmp + h2 * (-(ymp + zmp));
        Vnext(y) = (ymp + h2 * Vnext(x)) / (1 - P(a) * h2);
        Vnext(z) = (zmp + h2 * P(b)) / (1 - (Vnext(x) - P(c)) * h2);
    }
}