#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "thomas.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { b, stepsize, symmetry, method };
    enum methods { ExplicitEuler, ExplicitMidpoint, ExplicitRungeKutta4, VariableSymmetryCD};
    enum maps { LLE, MAX, Period };
}

__global__ void kernelProgram_thomas(Computation* data)
{
    int variation = (blockIdx.x * blockDim.x) + threadIdx.x;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int stepStart, variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    numb variables[MAX_ATTRIBUTES];
    numb variablesNext[MAX_ATTRIBUTES];
    numb parameters[MAX_ATTRIBUTES];
    LOAD_ATTRIBUTES;

    // Custom area (usually) starts here

    TRANSIENT_SKIP_NEW(finiteDifferenceScheme_thomas);

    for (int s = 0; s < CUDA_kernel.steps && !data->isHires; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;
        finiteDifferenceScheme_thomas(FDS_ARGUMENTS);
        RECORD_STEP;
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_thomas, MO(LLE));
    }

    if (M(MAX).toCompute)
    {
        MAX_Settings max_settings(MS(MAX, 0));
        MAX(data, max_settings, variation, &finiteDifferenceScheme_thomas, MO(MAX));
    }

    if (M(Period).toCompute)
    {
        DBscan_Settings dbscan_settings(MS(Period, 0), MS(Period, 1), MS(Period, 2), MS(Period, 3), MS(Period, 4), MS(Period, 5), MS(Period, 6), MS(Period, 7), attributes::parameters::stepsize);
        Period(data, dbscan_settings, variation, &finiteDifferenceScheme_thomas, MO(Period));
    }
}

__device__ __forceinline__ void finiteDifferenceScheme_thomas(numb* currentV, numb* nextV, numb* parameters)
{
    ifMETHOD(P(method), ExplicitEuler)
    {
        Vnext(x) = V(x) + P(stepsize) * (sin(V(y)) - P(b) * V(x));
        Vnext(y) = V(y) + P(stepsize) * (sin(V(z)) - P(b) * V(y));
        Vnext(z) = V(z) + P(stepsize) * (sin(V(x)) - P(b) * V(z));
    }

    ifMETHOD(P(method), ExplicitMidpoint)
    {
        numb xmp = V(x) + P(stepsize) * 0.5 * (sin(V(y)) - P(b) * V(x));
        numb ymp = V(y) + P(stepsize) * 0.5 * (sin(V(z)) - P(b) * V(y));
        numb zmp = V(z) + P(stepsize) * 0.5 * (sin(V(x)) - P(b) * V(z));

        Vnext(x) = V(x) + P(stepsize) * (sin(ymp) - P(b) * xmp);
        Vnext(y) = V(y) + P(stepsize) * (sin(zmp) - P(b) * ymp);
        Vnext(z) = V(z) + P(stepsize) * (sin(xmp) - P(b) * zmp);
    }

    ifMETHOD(P(method), ExplicitRungeKutta4)
    {
        numb kx1 = sin(V(y)) - P(b) * V(x);
        numb ky1 = sin(V(z)) - P(b) * V(y);
        numb kz1 = sin(V(x)) - P(b) * V(z);

        numb xmp = V(x) + 0.5 * P(stepsize) * kx1;
        numb ymp = V(y) + 0.5 * P(stepsize) * ky1;
        numb zmp = V(z) + 0.5 * P(stepsize) * kz1;

        numb kx2 = sin(ymp) - P(b) * xmp;
        numb ky2 = sin(zmp) - P(b) * ymp;
        numb kz2 = sin(xmp) - P(b) * zmp;

        xmp = V(x) + 0.5 * P(stepsize) * kx2;
        ymp = V(y) + 0.5 * P(stepsize) * ky2;
        zmp = V(z) + 0.5 * P(stepsize) * kz2;

        numb kx3 = sin(ymp) - P(b) * xmp;
        numb ky3 = sin(zmp) - P(b) * ymp;
        numb kz3 = sin(xmp) - P(b) * zmp;

        xmp = V(x) + P(stepsize) * kx3;
        ymp = V(y) + P(stepsize) * ky3;
        zmp = V(z) + P(stepsize) * kz3;

        numb kx4 = sin(ymp) - P(b) * xmp;
        numb ky4 = sin(zmp) - P(b) * ymp;
        numb kz4 = sin(xmp) - P(b) * zmp;

        Vnext(x) = V(x) + P(stepsize) * (kx1 + 2.0 * kx2 + 2.0 * kx3 + kx4) / 6.0;
        Vnext(y) = V(y) + P(stepsize) * (ky1 + 2.0 * ky2 + 2.0 * ky3 + ky4) / 6.0;
        Vnext(z) = V(z) + P(stepsize) * (kz1 + 2.0 * kz2 + 2.0 * kz3 + kz4) / 6.0;
    }
    ifMETHOD(P(method), VariableSymmetryCD)
    {  
        numb h1 = 0.5 * P(stepsize) - P(symmetry);
        numb h2 = 0.5 * P(stepsize) + P(symmetry);

        numb xmp = V(x) + h1 * (sin(V(y)) - P(b) * V(x));
        numb ymp = V(y) + h1 * (sin(V(z)) - P(b) * V(y));
        numb zmp = V(z) + h1 * (sin(xmp) - P(b) * V(z));

        numb denom_z = (1 + h2 * P(b));
        if (fabs(denom_z) < 1e-6) denom_z = copysign(1e-6, denom_z);
        Vnext(z) = (zmp + h2 * sin(xmp)) / denom_z;

        numb denom_y = (1 + h2 * P(b));
        if (fabs(denom_y) < 1e-6) denom_y = copysign(1e-6, denom_y);
        Vnext(y) = (ymp + h2 * sin(Vnext(z))) / denom_y;

        numb denom_x = (1 + h2 * P(b));
        if (fabs(denom_x) < 1e-6) denom_x = copysign(1e-6, denom_x);
        Vnext(x) = (xmp + h2 * sin(Vnext(y))) / denom_x;
    }
}