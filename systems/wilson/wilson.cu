#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "wilson.h"

namespace attributes
{
    enum variables { v, r, i, t };
    enum parameters { C, tau, p0, p1, p2, p3, p4, p5, p6, p7, Idc, Iamp, Ifreq, Idel, Idf, signal, method, COUNT };
    enum waveforms { square, sine, triangle };
    enum methods { ExplicitEuler, ExplicitMidpoint };
    enum maps { LLE, MAX, MeanInterval, MeanPeak, Period };
}

__global__ void kernelProgram_wilson(Computation* data)
{
    int variation = (blockIdx.x * blockDim.x) + threadIdx.x;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int stepStart, variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    numb variables[MAX_ATTRIBUTES];
    numb variablesNext[MAX_ATTRIBUTES];
    numb parameters[MAX_ATTRIBUTES];
    LOAD_ATTRIBUTES(false);

    // Custom area (usually) starts here

    TRANSIENT_SKIP_NEW(finiteDifferenceScheme_wilson);

    for (int s = 0; s < CUDA_kernel.steps && !data->isHires; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;
        finiteDifferenceScheme_wilson(FDS_ARGUMENTS);
        RECORD_STEP;
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_wilson, MO(LLE));
    }

    if (M(MAX).toCompute)
    {
        MAX_Settings max_settings(MS(MAX, 0));
        MAX(data, max_settings, variation, &finiteDifferenceScheme_wilson, MO(MAX));
    }

    if (M(Period).toCompute || M(MeanInterval).toCompute || M(MeanPeak).toCompute)
    {
        DBscan_Settings dbscan_settings(MS(Period, 0), MS(MeanInterval, 0), MS(Period, 1), MS(Period, 2), MS(MeanInterval, 1), MS(MeanInterval, 2), MS(MeanInterval, 3), MS(MeanInterval, 4),
            H);
        Period(data, dbscan_settings, variation, &finiteDifferenceScheme_wilson, MO(Period), MO(MeanPeak), MO(MeanInterval));
    }
}

__device__ __forceinline__ void finiteDifferenceScheme_wilson(numb* currentV, numb* nextV, numb* parameters)
{
    ifSIGNAL(P(signal), square)
    {
        ifMETHOD(P(method), ExplicitEuler)
        {
            Vnext(i) = P(Idc) + (fmodf((V(t) - P(Idel)) > 0 ? (V(t) - P(Idel)) : (P(Idf) / P(Ifreq) + P(Idel) - V(t)), 1 / P(Ifreq)) < P(Idf) / P(Ifreq) ? P(Iamp) : 0.0f);
            Vnext(t) = V(t) + H;
            Vnext(v) = V(v) + H * ((-(P(p0) + P(p1) * V(v) + P(p2) * V(v) * V(v)) * (V(v) - P(p3)) - P(p5) * V(r) * (V(v) - P(p4)) + Vnext(i)) / P(C));
            Vnext(r) = V(r) + H * ((1.0 / P(tau)) * (-V(r) + P(p6) * V(v) + P(p7)));
        }
        ifMETHOD(P(method), ExplicitMidpoint)
        {
            numb imp = P(Idc) + (fmodf((V(t) - P(Idel)) > 0 ? (V(t) - P(Idel)) : (P(Idf) / P(Ifreq) + P(Idel) - V(t)), 1 / P(Ifreq)) < P(Idf) / P(Ifreq) ? P(Iamp) : 0.0f);
            numb tmp = V(t) + H * 0.5;
            numb vmp = V(v) + H * 0.5 * ((-(P(p0) + P(p1) * V(v) + P(p2) * V(v) * V(v)) * (V(v) - P(p3)) - P(p5) * V(r) * (V(v) - P(p4)) + imp) / P(C));
            numb rmp = V(r) + H * 0.5 * ((1.0 / P(tau)) * (-V(r) + P(p6) * V(v) + P(p7)));

            Vnext(i) = P(Idc) + (fmodf((tmp - P(Idel)) > 0 ? (tmp - P(Idel)) : (P(Idf) / P(Ifreq) + P(Idel) - tmp), 1 / P(Ifreq)) < P(Idf) / P(Ifreq) ? P(Iamp) : 0.0f);
            Vnext(t) = V(t) + H;
            Vnext(v) = V(v) + H * ((-(P(p0) + P(p1) * vmp + P(p2) * vmp * vmp) * (vmp - P(p3)) - P(p5) * rmp * (vmp - P(p4)) + Vnext(i)) / P(C));
            Vnext(r) = V(r) + H * ((1.0 / P(tau)) * (-rmp + P(p6) * vmp + P(p7)));
        }
    }
    ifSIGNAL(P(signal), sine)
    {
        ifMETHOD(P(method), ExplicitEuler)
        {
            Vnext(i) = P(Idc) + P(Iamp) * sinf(2.0f * 3.141592f * P(Ifreq) * (V(t) - P(Idel)));
            Vnext(t) = V(t) + H;
            Vnext(v) = V(v) + H * ((-(P(p0) + P(p1) * V(v) + P(p2) * V(v) * V(v)) * (V(v) - P(p3)) - P(p5) * V(r) * (V(v) - P(p4)) + Vnext(i)) / P(C));
            Vnext(r) = V(r) + H * ((1.0 / P(tau)) * (-V(r) + P(p6) * V(v) + P(p7)));
        }
        ifMETHOD(P(method), ExplicitMidpoint)
        {
            numb imp = P(Idc) + P(Iamp) * sinf(2.0f * 3.141592f * P(Ifreq) * (V(t) - P(Idel)));
            numb tmp = V(t) + H * 0.5;
            numb vmp = V(v) + H * 0.5 * ((-(P(p0) + P(p1) * V(v) + P(p2) * V(v) * V(v)) * (V(v) - P(p3)) - P(p5) * V(r) * (V(v) - P(p4)) + imp) / P(C));
            numb rmp = V(r) + H * 0.5 * ((1.0 / P(tau)) * (-V(r) + P(p6) * V(v) + P(p7)));

            Vnext(i) = P(Idc) + P(Iamp) * sinf(2.0f * 3.141592f * P(Ifreq) * (tmp - P(Idel)));
            Vnext(t) = V(t) + H;
            Vnext(v) = V(v) + H * ((-(P(p0) + P(p1) * vmp + P(p2) * vmp * vmp) * (vmp - P(p3)) - P(p5) * rmp * (vmp - P(p4)) + Vnext(i)) / P(C));
            Vnext(r) = V(r) + H * ((1.0 / P(tau)) * (-rmp + P(p6) * vmp + P(p7)));
        }
    }
    ifSIGNAL(P(signal), triangle)
    {
        ifMETHOD(P(method), ExplicitEuler)
        {
            Vnext(i) = P(Idc) + P(Iamp) * ((4 * P(Ifreq) * (V(t) - P(Idel)) - 2 * floorf((4 * P(Ifreq) * (V(t) - P(Idel)) + 1) / 2)) * pow((-1), floorf((4 * P(Ifreq) * (V(t) - P(Idel)) + 1) / 2)));
            Vnext(t) = V(t) + H;
            Vnext(v) = V(v) + H * ((-(P(p0) + P(p1) * V(v) + P(p2) * V(v) * V(v)) * (V(v) - P(p3)) - P(p5) * V(r) * (V(v) - P(p4)) + Vnext(i)) / P(C));
            Vnext(r) = V(r) + H * ((1.0 / P(tau)) * (-V(r) + P(p6) * V(v) + P(p7)));
        }
        ifMETHOD(P(method), ExplicitMidpoint)
        {
            numb imp = P(Idc) + P(Iamp) * ((4 * P(Ifreq) * (V(t) - P(Idel)) - 2 * floorf((4 * P(Ifreq) * (V(t) - P(Idel)) + 1) / 2)) * pow((-1), floorf((4 * P(Ifreq) * (V(t) - P(Idel)) + 1) / 2)));
            numb tmp = V(t) + H * 0.5;
            numb vmp = V(v) + H * 0.5 * ((-(P(p0) + P(p1) * V(v) + P(p2) * V(v) * V(v)) * (V(v) - P(p3)) - P(p5) * V(r) * (V(v) - P(p4)) + imp) / P(C));
            numb rmp = V(r) + H * 0.5 * ((1.0 / P(tau)) * (-V(r) + P(p6) * V(v) + P(p7)));

            Vnext(i) = P(Idc) + P(Iamp) * ((4 * P(Ifreq) * (tmp - P(Idel)) - 2 * floorf((4 * P(Ifreq) * (tmp - P(Idel)) + 1) / 2)) * pow((-1), floorf((4 * P(Ifreq) * (tmp - P(Idel)) + 1) / 2)));
            Vnext(t) = V(t) + H;
            Vnext(v) = V(v) + H * ((-(P(p0) + P(p1) * vmp + P(p2) * vmp * vmp) * (vmp - P(p3)) - P(p5) * rmp * (vmp - P(p4)) + Vnext(i)) / P(C));
            Vnext(r) = V(r) + H * ((1.0 / P(tau)) * (-rmp + P(p6) * vmp + P(p7)));
        }
    }
}