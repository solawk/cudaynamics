#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "lorenz.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { sigma, rho, beta, stepsize, symmetry, method };
    enum methods { ExplicitEuler, ExplicitMidpoint, ExplicitRungeKutta4, VariableSymmetryCD };
    enum maps { LLE, MAX };
}

__global__ void kernelProgram_lorenz(Computation* data)
{
    int b = blockIdx.x;                                     // Current block of THREADS_PER_BLOCK threads
    int t = threadIdx.x;                                    // Current thread in the block, from 0 to THREADS_PER_BLOCK-1
    int variation = (b * data->threads_per_block) + t;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    int stepStart = variationStart;                         // Start index for the current modelling step

    // Custom area (usually) starts here

    TRANSIENT_SKIP(finiteDifferenceScheme_lorenz);

    for (int s = 0; s < CUDA_kernel.steps; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;

        finiteDifferenceScheme_lorenz(&(CUDA_marshal.trajectory[stepStart]),
            &(CUDA_marshal.trajectory[stepStart + CUDA_kernel.VAR_COUNT]),
            &(CUDA_marshal.parameterVariations[variation * CUDA_kernel.PARAM_COUNT]));
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_lorenz, MO(LLE));
    }

    if (M(MAX).toCompute)
    {
        MAX_Settings max_settings(MS(MAX, 0));
        MAX(data, max_settings, variation, &finiteDifferenceScheme_lorenz, MO(MAX));
    }
}

__device__ void finiteDifferenceScheme_lorenz(numb* currentV, numb* nextV, numb* parameters)
{
    ifMETHOD(P(method), ExplicitEuler)
    {
        Vnext(x) = V(x) + P(stepsize) * (P(sigma) * (V(y) - V(x)));
        Vnext(y) = V(y) + P(stepsize) * (V(x) * (P(rho) - V(z)) - V(y));
        Vnext(z) = V(z) + P(stepsize) * (V(x) * V(y) - P(beta) * V(z));
    }

    ifMETHOD(P(method), ExplicitMidpoint)
    {
        numb xmp = V(x) + P(stepsize) * 0.5 * (P(sigma) * (V(y) - V(x)));
        numb ymp = V(y) + P(stepsize) * 0.5 * (V(x) * (P(rho) - V(z)) - V(y));
        numb zmp = V(z) + P(stepsize) * 0.5 * (V(x) * V(y) - P(beta) * V(z));

        Vnext(x) = V(x) + P(stepsize) * (P(sigma) * (ymp - xmp));
        Vnext(y) = V(y) + P(stepsize) * (xmp * (P(rho) - zmp) - ymp);
        Vnext(z) = V(z) + P(stepsize) * (xmp * ymp - P(beta) * zmp);
    }

    ifMETHOD(P(method), ExplicitRungeKutta4)
    {
        numb kx1 = P(sigma) * (V(y) - V(x));
        numb ky1 = V(x) * (P(rho) - V(z)) - V(y);
        numb kz1 = V(x) * V(y) - P(beta) * V(z);

        numb xmp = V(x) + 0.5 * P(stepsize) * kx1;
        numb ymp = V(y) + 0.5 * P(stepsize) * ky1;
        numb zmp = V(z) + 0.5 * P(stepsize) * kz1;

        numb kx2 = P(sigma) * (ymp - xmp);
        numb ky2 = xmp * (P(rho) - zmp) - ymp;
        numb kz2 = xmp * ymp - P(beta) * zmp;

        xmp = V(x) + 0.5 * P(stepsize) * kx2;
        ymp = V(y) + 0.5 * P(stepsize) * ky2;
        zmp = V(z) + 0.5 * P(stepsize) * kz2;

        numb kx3 = P(sigma) * (ymp - xmp);
        numb ky3 = xmp * (P(rho) - zmp) - ymp;
        numb kz3 = xmp * ymp - P(beta) * zmp;

        xmp = V(x) + P(stepsize) * kx3;
        ymp = V(y) + P(stepsize) * ky3;
        zmp = V(z) + P(stepsize) * kz3;

        numb kx4 = P(sigma) * (ymp - xmp);
        numb ky4 = xmp * (P(rho) - zmp) - ymp;
        numb kz4 = xmp * ymp - P(beta) * zmp;

        Vnext(x) = V(x) + P(stepsize) * (kx1 + 2 * kx2 + 2 * kx3 + kx4) / 6;
        Vnext(y) = V(y) + P(stepsize) * (ky1 + 2 * ky2 + 2 * ky3 + ky4) / 6;
        Vnext(z) = V(z) + P(stepsize) * (kz1 + 2 * kz2 + 2 * kz3 + kz4) / 6;
    }

    ifMETHOD(P(method), VariableSymmetryCD)
    {
        numb h1 = 0.5 * P(stepsize) - P(symmetry);
        numb h2 = 0.5 * P(stepsize) + P(symmetry);

        numb xmp = V(x) + h1 * (P(sigma) * (V(y) - V(x)));
        numb ymp = V(y) + h1 * (V(x) * (P(rho) - V(z)) - V(y));
        numb zmp = V(z) + h1 * (V(x) * V(y) - P(beta) * V(z));

        Vnext(z) = (zmp + xmp * ymp * h2) / (1 + P(beta) * h2);
        Vnext(y) = (ymp + xmp * (P(rho) - Vnext(z)) * h2) / (1 + h2);
        Vnext(x) = (xmp + P(sigma) * Vnext(y) * h2) / (1 + P(sigma) * h2);
    }
}