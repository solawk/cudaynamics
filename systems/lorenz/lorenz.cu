#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "lorenz.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { sigma, rho, beta, symmetry, method };
    enum methods { ExplicitEuler, ExplicitMidpoint, ExplicitRungeKutta4, VariableSymmetryCD };
    enum maps { LLE, MAX, MeanInterval, MeanPeak, Period };
}

__global__ void kernelProgram_lorenz(Computation* data)
{
    int variation = (blockIdx.x * blockDim.x) + threadIdx.x;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int stepStart, variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    LOCAL_BUFFERS;
    LOAD_ATTRIBUTES;

    // Custom area (usually) starts here

    TRANSIENT_SKIP_NEW(finiteDifferenceScheme_lorenz);

    for (int s = 0; s < CUDA_kernel.steps && !data->isHires; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;
        finiteDifferenceScheme_lorenz(FDS_ARGUMENTS);
        RECORD_STEP;
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_lorenz, MO(LLE));
    }

    if (M(MAX).toCompute)
    {
        MAX_Settings max_settings(MS(MAX, 0));
        MAX(data, max_settings, variation, &finiteDifferenceScheme_lorenz, MO(MAX));
    }

    if (M(Period).toCompute || M(MeanInterval).toCompute || M(MeanPeak).toCompute)
    {
        DBscan_Settings dbscan_settings(MS(Period, 0), MS(MeanInterval, 0), MS(Period, 1), MS(Period, 2), MS(MeanInterval, 1), MS(MeanInterval, 2), MS(MeanInterval, 3), MS(MeanInterval, 4),
            H_BRANCH(parameters[CUDA_kernel.PARAM_COUNT - 1], variables[CUDA_kernel.VAR_COUNT - 1]));
        Period(data, dbscan_settings, variation, &finiteDifferenceScheme_lorenz, MO(Period), MO(MeanPeak), MO(MeanInterval));
    }
}

__device__ __forceinline__ void finiteDifferenceScheme_lorenz(numb* currentV, numb* nextV, numb* parameters, Computation* data)
{
    ifMETHOD(P(method), ExplicitEuler)
    {
        Vnext(x) = V(x) + H * (P(sigma) * (V(y) - V(x)));
        Vnext(y) = V(y) + H * (V(x) * (P(rho) - V(z)) - V(y));
        Vnext(z) = V(z) + H * (V(x) * V(y) - P(beta) * V(z));
    }

    ifMETHOD(P(method), ExplicitMidpoint)
    {
        numb xmp = V(x) + H * 0.5f * (P(sigma) * (V(y) - V(x)));
        numb ymp = V(y) + H * 0.5f * (V(x) * (P(rho) - V(z)) - V(y));
        numb zmp = V(z) + H * 0.5f * (V(x) * V(y) - P(beta) * V(z));

        Vnext(x) = V(x) + H * (P(sigma) * (ymp - xmp));
        Vnext(y) = V(y) + H * (xmp * (P(rho) - zmp) - ymp);
        Vnext(z) = V(z) + H * (xmp * ymp - P(beta) * zmp);
    }

    ifMETHOD(P(method), ExplicitRungeKutta4)
    {
        numb kx1 = P(sigma) * (V(y) - V(x));
        numb ky1 = V(x) * (P(rho) - V(z)) - V(y);
        numb kz1 = V(x) * V(y) - P(beta) * V(z);

        numb xmp = V(x) + 0.5f * H * kx1;
        numb ymp = V(y) + 0.5f * H * ky1;
        numb zmp = V(z) + 0.5f * H * kz1;

        numb kx2 = P(sigma) * (ymp - xmp);
        numb ky2 = xmp * (P(rho) - zmp) - ymp;
        numb kz2 = xmp * ymp - P(beta) * zmp;

        xmp = V(x) + 0.5f * H * kx2;
        ymp = V(y) + 0.5f * H * ky2;
        zmp = V(z) + 0.5f * H * kz2;

        numb kx3 = P(sigma) * (ymp - xmp);
        numb ky3 = xmp * (P(rho) - zmp) - ymp;
        numb kz3 = xmp * ymp - P(beta) * zmp;

        xmp = V(x) + H * kx3;
        ymp = V(y) + H * ky3;
        zmp = V(z) + H * kz3;

        numb kx4 = P(sigma) * (ymp - xmp);
        numb ky4 = xmp * (P(rho) - zmp) - ymp;
        numb kz4 = xmp * ymp - P(beta) * zmp;

        Vnext(x) = V(x) + H * (kx1 + 2.0f * kx2 + 2.0f * kx3 + kx4) / 6.0f;
        Vnext(y) = V(y) + H * (ky1 + 2.0f * ky2 + 2.0f * ky3 + ky4) / 6.0f;
        Vnext(z) = V(z) + H * (kz1 + 2.0f * kz2 + 2.0f * kz3 + kz4) / 6.0f;
    }

    ifMETHOD(P(method), VariableSymmetryCD)
    {
        numb h1 = 0.5f * H - P(symmetry);
        numb h2 = 0.5f * H + P(symmetry);

        numb xmp = V(x) + h1 * (P(sigma) * (V(y) - V(x)));
        numb ymp = V(y) + h1 * (xmp * (P(rho) - V(z)) - V(y));
        numb zmp = V(z) + h1 * (xmp * ymp - P(beta) * V(z));

        Vnext(z) = (zmp + xmp * ymp * h2) / (1.0f + P(beta) * h2);
        Vnext(y) = (ymp + xmp * (P(rho) - Vnext(z)) * h2) / (1.0f + h2);
        Vnext(x) = (xmp + P(sigma) * Vnext(y) * h2) / (1.0f + P(sigma) * h2);
    }
}