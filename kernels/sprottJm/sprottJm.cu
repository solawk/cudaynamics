#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "sprottJm.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { a, b, c, stepsize, method };
    enum methods { ExplicitEuler, ExplicitRungeKutta4 };
    enum maps { LLE };
}

__global__ void kernelProgram_sprottJm(Computation* data)
{
    int b = blockIdx.x;                                     // Current block of THREADS_PER_BLOCK threads
    int t = threadIdx.x;                                    // Current thread in the block, from 0 to THREADS_PER_BLOCK-1
    int variation = (b * data->threads_per_block) + t;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    int stepStart = variationStart;                         // Start index for the current modelling step

    // Custom area (usually) starts here

    TRANSIENT_SKIP(finiteDifferenceScheme_sprottJm);

    for (int s = 0; s < CUDA_kernel.steps; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;

        finiteDifferenceScheme_sprottJm(&(CUDA_marshal.trajectory[stepStart]),
            &(CUDA_marshal.trajectory[stepStart + CUDA_kernel.VAR_COUNT]),
            &(CUDA_marshal.parameterVariations[variation * CUDA_kernel.PARAM_COUNT]),
            CUDA_kernel.stepSize);
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_sprottJm, MO(LLE));
    }
}

__device__ numb sprottJm_F(numb y)
{
    numb b, k;
    numb d = 1.0, m = 1.0, P = 1.23, R = 2.0;
    numb ay = abs(y);
    while (true)
    {
        if (ay < d)
        {
            d /= R; m /= R;
        }
        else if (ay > 2 * d)
        {
            d *= R; m *= R;
        }
        else
            break;
    }
    numb epsilon = 0.01;
    if (d > epsilon)
    {
        if (ay < P * d)
        {
            b = -m * (R - P * R + 1) / (R * (P - 1));
            k = -m / (R * d * (1 - P));
        }
        else
        {
            b = -m * (R - P * R + 1) / (P - R);
            k = -m * (-(R * R) + R + 1) / (R * d * (P - R));
        }

        return k * ay + b;
    }
    else
        return 0.0;
}

__device__ void finiteDifferenceScheme_sprottJm(numb* currentV, numb* nextV, numb* parameters, numb h)
{  
    ifMETHOD(P(method), ExplicitEuler)
    {
        Vnext(x) = V(x) + P(stepsize) * (P(a) * V(z));
        Vnext(y) = V(y) + P(stepsize) * (P(b) * V(y) + V(z));
        Vnext(z) = V(z) + P(stepsize) * (-V(x) + V(y) + P(c) * sprottJm_F(V(y)));
    }

    ifMETHOD(P(method), ExplicitRungeKutta4)
    {
        numb kx1 = P(a) * V(z);
        numb ky1 = P(b) * V(y) + V(z);
        numb kz1 = -V(x) + V(y) + P(c) * sprottJm_F(V(y));

        numb xmp = V(x) + 0.5 * P(stepsize) * kx1;
        numb ymp = V(y) + 0.5 * P(stepsize) * ky1;
        numb zmp = V(z) + 0.5 * P(stepsize) * kz1;

        numb kx2 = P(a) * zmp;
        numb ky2 = P(b) * ymp + zmp;
        numb kz2 = -xmp + ymp + P(c) * sprottJm_F(ymp);

        xmp = V(x) + 0.5 * P(stepsize) * kx2;
        ymp = V(y) + 0.5 * P(stepsize) * ky2;
        zmp = V(z) + 0.5 * P(stepsize) * kz2;

        numb kx3 = P(a) * zmp;
        numb ky3 = P(b) * ymp + zmp;
        numb kz3 = -xmp + ymp + P(c) * sprottJm_F(ymp);

        xmp = V(x) + P(stepsize) * kx3;
        ymp = V(y) + P(stepsize) * ky3;
        zmp = V(z) + P(stepsize) * kz3;

        numb kx4 = P(a) * zmp;
        numb ky4 = P(b) * ymp + zmp;
        numb kz4 = -xmp + ymp + P(c) * sprottJm_F(ymp);

        Vnext(x) = V(x) + P(stepsize) * (kx1 + 2 * kx2 + 2 * kx3 + kx4) / 6;
        Vnext(y) = V(y) + P(stepsize) * (ky1 + 2 * ky2 + 2 * ky3 + ky4) / 6;
        Vnext(z) = V(z) + P(stepsize) * (kz1 + 2 * kz2 + 2 * kz3 + kz4) / 6;
    }
}