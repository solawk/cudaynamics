#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "lorenz2.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { sigma, rho, beta };
    enum maps { LLE };
}

__global__ void kernelProgram_lorenz2(Computation* data)
{
    int b = blockIdx.x;                                     // Current block of THREADS_PER_BLOCK threads
    int t = threadIdx.x;                                    // Current thread in the block, from 0 to THREADS_PER_BLOCK-1
    int variation = (b * data->threads_per_block) + t;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    int stepStart = variationStart;                         // Start index for the current modelling step
    //int indicesStart = variation * (CUDA_kernel.VAR_COUNT + CUDA_kernel.PARAM_COUNT);   // Start index for the step indices of the attributes in the current variation

    // Custom area (usually) starts here

    TRANSIENT_SKIP(finiteDifferenceScheme_lorenz2);

    for (int s = 0; s < CUDA_kernel.steps; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;

        finiteDifferenceScheme_lorenz2(&(CUDA_marshal.trajectory[stepStart]),
            &(CUDA_marshal.trajectory[stepStart + CUDA_kernel.VAR_COUNT]),
            &(CUDA_marshal.parameterVariations[variation * CUDA_kernel.PARAM_COUNT]),
            CUDA_kernel.stepSize);
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(0.01f, 50, 0);
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_lorenz2);
    }
}

__device__ void finiteDifferenceScheme_lorenz2(numb* currentV, numb* nextV, numb* parameters, numb h)
{
    numb dx = P(sigma) * (V(y) - V(x));
    numb dy = V(x) * (P(rho) - V(z)) - V(y);
    numb dz = V(x) * V(y) - P(beta) * V(z);

    Vnext(x) = V(x) + h * dx;
    Vnext(y) = V(y) + h * dy;
    Vnext(z) = V(z) + h * dz;
}