#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "fourwing.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { a, b, c, symmetry, method };
    enum methods { ExplicitEuler, ExplicitMidpoint, ExplicitRungeKutta4, VariableSymmetryCD};
    enum maps { LLE };
}

__global__ void kernelProgram_fourwing(Computation* data)
{
    int b = blockIdx.x;                                     // Current block of THREADS_PER_BLOCK threads
    int t = threadIdx.x;                                    // Current thread in the block, from 0 to THREADS_PER_BLOCK-1
    int variation = (b * data->threads_per_block) + t;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    int stepStart = variationStart;                         // Start index for the current modelling step

    // Custom area (usually) starts here

    TRANSIENT_SKIP(finiteDifferenceScheme_fourwing);

    for (int s = 0; s < CUDA_kernel.steps; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;

        finiteDifferenceScheme_fourwing(&(CUDA_marshal.trajectory[stepStart]),
            &(CUDA_marshal.trajectory[stepStart + CUDA_kernel.VAR_COUNT]),
            &(CUDA_marshal.parameterVariations[variation * CUDA_kernel.PARAM_COUNT]),
            CUDA_kernel.stepSize);
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_fourwing, MO(LLE));
    }
}

__device__ void finiteDifferenceScheme_fourwing(numb* currentV, numb* nextV, numb* parameters, numb h)
{
    ifMETHOD(P(method), ExplicitEuler)
    {
        numb dx = P(a)*V(x) + V(y)*V(z);
        numb dy = P(b)*V(x) + P(c)*V(y) - V(x)*V(z);
        numb dz = -V(z) - V(x)*V(y);

        Vnext(x) = V(x) + h * dx;
        Vnext(y) = V(y) + h * dy;
        Vnext(z) = V(z) + h * dz;
    }

    ifMETHOD(P(method), ExplicitMidpoint)
    {
        numb dx = P(a)*V(x) + V(y)*V(z);
        numb dy = P(b)*V(x) + P(c)*V(y) - V(x)*V(z);
        numb dz = -V(z) - V(x)*V(y);

        numb xmp = V(x) + h * 0.5 * dx;
        numb ymp = V(y) + h * 0.5 * dy;
        numb zmp = V(z) + h * 0.5 * dz;

        numb dx2 = P(a)*xmp + ymp*zmp;
        numb dy2 = P(b)*xmp + P(c)*ymp - xmp*zmp;
        numb dz2 = -zmp - xmp*ymp;

        Vnext(x) = V(x) + h * dx2;
        Vnext(y) = V(y) + h * dy2;
        Vnext(z) = V(z) + h * dz2;
    }

    ifMETHOD(P(method), ExplicitRungeKutta4)
    {
        

        numb dx1 = P(a) * V(x) + V(y) * V(z);
        numb dy1 = P(b) * V(x) + P(c) * V(y) - V(x) * V(z);
        numb dz1 = -V(z) - V(x) * V(y);

        numb xmp = V(x) + 0.5 * h * dx1;
        numb ymp = V(y) + 0.5 * h * dy1;
        numb zmp = V(z) + 0.5 * h * dz1;

        numb dx2 = P(a) * xmp + ymp * zmp;
        numb dy2 = P(b) * xmp + P(c) * ymp - V(x) * zmp;
        numb dz2 = -zmp - xmp * ymp;

        xmp = V(x) + 0.5 * h * dx2;
        ymp = V(y) + 0.5 * h * dy2;
        zmp = V(z) + 0.5 * h * dz2;

        numb dx3 = P(a) * xmp + ymp * zmp;
        numb dy3 = P(b) * xmp + P(c) * ymp - V(x) * zmp;
        numb dz3 = -zmp - xmp * ymp;

        xmp = V(x) + h * dx3;
        ymp = V(y) + h * dy3;
        zmp = V(z) + h * dz3;

        numb dx4 = P(a) * xmp + ymp * zmp;
        numb dy4 = P(b) * xmp + P(c) * ymp - V(x) * zmp;
        numb dz4 = -zmp - xmp * ymp;

        Vnext(x) = V(x) + h * (dx1 + 2 * dx2 + 2 * dx3 + dx4) / 6;
        Vnext(y) = V(y) + h * (dy1 + 2 * dy2 + 2 * dy3 + dy4) / 6;
        Vnext(z) = V(z) + h * (dz1 + 2 * dz2 + 2 * dz3 + dz4) / 6;

    }

    ifMETHOD(P(method), VariableSymmetryCD)
    {
        numb s = P(symmetry);
        numb h1 = 0.5 * h - s;
        numb h2 = 0.5 * h + s;

        numb dx1 = P(a) * V(x) + V(y) * V(z);
        numb xmp1 = V(x) + h1 * dx1;
        numb dy1 = P(b) * xmp1 + P(c) * V(y) - xmp1 * V(z);
        numb ymp1 = V(y) + h1 * dy1;
        numb dz1 = -V(z) - xmp1 * ymp1;
        numb zmp1 = V(z) + h1 * dz1;

        numb xmp2 = xmp1;
        numb ymp2 = ymp1;
        numb zmp2 = zmp1;

        zmp1 = (zmp2-h2*xmp1*ymp1)/(1+h2);
        ymp1 = (ymp2+h2*P(b)*xmp1-h2*xmp1*zmp1)/(1-h2*P(c));
        xmp1 = (xmp2+ymp1*zmp1*h2)/(1-P(a)*h2);

        Vnext(x) = xmp1;
        Vnext(y) = ymp1;
        Vnext(z) = zmp1;

    }
}