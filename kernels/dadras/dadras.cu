#include "hip/hip_runtime.h"
#include "main.h"
#include "dadras.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { a, b, c, d, e, symmetry, method };
    enum methods { ExplicitEuler, ExplicitMidpoint, ExplicitRungeKutta4, VariableSymmetryCD};
    enum maps { LLE };
}

__global__ void kernelProgram_dadras(Computation* data)
{
    int b = blockIdx.x;                                     // Current block of THREADS_PER_BLOCK threads
    int t = threadIdx.x;                                    // Current thread in the block, from 0 to THREADS_PER_BLOCK-1
    int variation = (b * data->threads_per_block) + t;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    int stepStart = variationStart;                         // Start index for the current modelling step

    // Custom area (usually) starts here

    TRANSIENT_SKIP(finiteDifferenceScheme_dadras);

    for (int s = 0; s < CUDA_kernel.steps; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;

        finiteDifferenceScheme_dadras(&(CUDA_marshal.trajectory[stepStart]),
            &(CUDA_marshal.trajectory[stepStart + CUDA_kernel.VAR_COUNT]),
            &(CUDA_marshal.parameterVariations[variation * CUDA_kernel.PARAM_COUNT]),
            CUDA_kernel.stepSize);
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_dadras, MO(LLE));
    }
}

__device__ void finiteDifferenceScheme_dadras(numb* currentV, numb* nextV, numb* parameters, numb h)
{
    ifMETHOD(P(method), ExplicitEuler)
    {
        numb dx = V(y) + P(a)*V(x) + P(b)*V(y)*V(z);
        numb dy = P(c)*V(y) - V(x)*V(z) + V(z);
        numb dz = P(d)*V(x)*V(y) - P(e)*V(z);

        Vnext(x) = V(x) + h * dx;
        Vnext(y) = V(y) + h * dy;
        Vnext(z) = V(z) + h * dz;
    }

    ifMETHOD(P(method), ExplicitMidpoint)
    {
        numb dx = V(y) - P(a) * V(x) + P(b) * V(y) * V(z);
        numb dy = P(c) * V(y) - V(x) * V(z) + V(z);
        numb dz = P(d) * V(x) * V(y) - P(e) * V(z);

        numb xmp = V(x) + h * 0.5 * dx;
        numb ymp = V(y) + h * 0.5 * dy;
        numb zmp = V(z) + h * 0.5 * dz;

        numb dx2 = ymp - P(a) * xmp + P(b) * ymp * zmp;
        numb dy2 = P(c) * ymp - xmp * zmp + zmp;
        numb dz2 = P(d) * xmp * ymp - P(e) * zmp;

        Vnext(x) = V(x) + h * dx2;
        Vnext(y) = V(y) + h * dy2;
        Vnext(z) = V(z) + h * dz2;
    }

    ifMETHOD(P(method), ExplicitRungeKutta4)
    {


        numb dx1 = V(y) - P(a) * V(x) + P(b) * V(y) * V(z);
        numb dy1 = P(c) * V(y) - V(x) * V(z) + V(z);
        numb dz1 = P(d) * V(x) * V(y) - P(e) * V(z);

        numb xmp = V(x) + 0.5 * h * dx1;
        numb ymp = V(y) + 0.5 * h * dy1;
        numb zmp = V(z) + 0.5 * h * dz1;

        numb dx2 = ymp - P(a) * xmp + P(b) * ymp * zmp;
        numb dy2 = P(c) * ymp - xmp * zmp + zmp;
        numb dz2 = P(d) * xmp * ymp - P(e) * zmp;

        xmp = V(x) + 0.5 * h * dx2;
        ymp = V(y) + 0.5 * h * dy2;
        zmp = V(z) + 0.5 * h * dz2;

        numb dx3 = ymp - P(a) * xmp + P(b) * ymp * zmp;
        numb dy3 = P(c) * ymp - xmp * zmp + zmp;
        numb dz3 = P(d) * xmp * ymp - P(e) * zmp;

        xmp = V(x) + h * dx3;
        ymp = V(y) + h * dy3;
        zmp = V(z) + h * dz3;

        numb dx4 = ymp - P(a) * xmp + P(b) * ymp * zmp;
        numb dy4 = P(c) * ymp - xmp * zmp + zmp;
        numb dz4 = P(d) * xmp * ymp - P(e) * zmp;

        Vnext(x) = V(x) + h * (dx1 + 2 * dx2 + 2 * dx3 + dx4) / 6;
        Vnext(y) = V(y) + h * (dy1 + 2 * dy2 + 2 * dy3 + dy4) / 6;
        Vnext(z) = V(z) + h * (dz1 + 2 * dz2 + 2 * dz3 + dz4) / 6;

    }

    ifMETHOD(P(method), VariableSymmetryCD)
    {
        numb s = P(symmetry);

        numb h1 = 0.5*h -s;
        numb h2 = 0.5 * h + s;

        numb dx1 = V(y) - P(a) * V(x) + P(b) * V(y) * V(z);
        numb xmp1 = V(x) + h1 * dx1;
        numb dy1 = P(c) * V(y) - xmp1 * V(z) + V(z);
        numb ymp1 = V(y) + h1 * dy1;
        numb dz1 = P(d) * xmp1 * ymp1 - P(e) * V(z);
        numb zmp1 = V(z) + h1 * dz1;
        
        numb xmp2 = xmp1;
        numb ymp2 = ymp1;
        numb zmp2 = zmp1;

        zmp1 = (zmp2+P(d)*xmp1*ymp1*h2)/(1+P(e)*h2);
        ymp1 = (ymp2-xmp1*zmp1*h2+zmp1*h2)/(1-h2*P(c));
        xmp1 = (xmp2+h2*ymp1+P(b)*ymp1*zmp1*h2)/(1+P(a)*h2);

        Vnext(x) = xmp1;
        Vnext(y) = ymp1;
        Vnext(z) = zmp1;
    }


}