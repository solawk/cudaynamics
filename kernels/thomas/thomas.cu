#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "thomas.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { b, method };
	enum methods { ExplicitEuler, ExplicitMidpoint };
    enum maps { LLE };
}

__global__ void kernelProgram_thomas(Computation* data)
{
    int b = blockIdx.x;                                     // Current block of THREADS_PER_BLOCK threads
    int t = threadIdx.x;                                    // Current thread in the block, from 0 to THREADS_PER_BLOCK-1
    int variation = (b * data->threads_per_block) + t;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    int stepStart = variationStart;                         // Start index for the current modelling step

    // Custom area (usually) starts here

    TRANSIENT_SKIP(finiteDifferenceScheme_thomas);

    for (int s = 0; s < CUDA_kernel.steps; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;

        finiteDifferenceScheme_thomas(&(CUDA_marshal.trajectory[stepStart]),
            &(CUDA_marshal.trajectory[stepStart + CUDA_kernel.VAR_COUNT]),
            &(CUDA_marshal.parameterVariations[variation * CUDA_kernel.PARAM_COUNT]),
            CUDA_kernel.stepSize);
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_thomas);
    }
}

__device__ void finiteDifferenceScheme_thomas(numb* currentV, numb* nextV, numb* parameters, numb h)
{
    ifMETHOD(P(method), ExplicitEuler)
    {
        numb dx = sin(V(y)) - P(b) * V(x);
        numb dy = sin(V(z)) - P(b) * V(y);
        numb dz = sin(V(x)) - P(b) * V(z);

        Vnext(x) = V(x) + h * dx;
        Vnext(y) = V(y) + h * dy;
        Vnext(z) = V(z) + h * dz;
    }

    ifMETHOD(P(method), ExplicitMidpoint)
    {
        numb dx = sin(V(y)) - P(b) * V(x);
        numb dy = sin(V(z)) - P(b) * V(y);
        numb dz = sin(V(x)) - P(b) * V(z);

        numb xmp = V(x) + h * 0.5 * dx;
        numb ymp = V(y) + h * 0.5 * dy;
        numb zmp = V(z) + h * 0.5 * dz;

        numb dx2 = sin(ymp) - P(b) * xmp;
        numb dy2 = sin(zmp) - P(b) * ymp;
        numb dz2 = sin(xmp) - P(b) * zmp;

        Vnext(x) = V(x) + h * dx2;
        Vnext(y) = V(y) + h * dy2;
        Vnext(z) = V(z) + h * dz2;
    }
}