#include "hip/hip_runtime.h"
﻿#include "main.h"
#include "hindmarsh_rose.h"

namespace attributes
{
    enum variables { x, y, z };
    enum parameters { a, b, c, d, r, s, e, Iext, stepsize, symmetry, method };
    enum methods { ExplicitEuler, ExplicitMidpoint, ExplicitRungeKutta4, VariableSymmetryCD};
    enum maps { LLE };
}

__global__ void kernelProgram_hindmarsh_rose(Computation* data)
{
    int b = blockIdx.x;                                     // Current block of THREADS_PER_BLOCK threads
    int t = threadIdx.x;                                    // Current thread in the block, from 0 to THREADS_PER_BLOCK-1
    int variation = (b * data->threads_per_block) + t;            // Variation (parameter combination) index
    if (variation >= CUDA_marshal.totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int variationStart = variation * CUDA_marshal.variationSize;         // Start index to store the modelling data for the variation
    int stepStart = variationStart;                         // Start index for the current modelling step

    // Custom area (usually) starts here

    TRANSIENT_SKIP(finiteDifferenceScheme_hindmarsh_rose);

    for (int s = 0; s < CUDA_kernel.steps; s++)
    {
        stepStart = variationStart + s * CUDA_kernel.VAR_COUNT;

        finiteDifferenceScheme_hindmarsh_rose(&(CUDA_marshal.trajectory[stepStart]),
            &(CUDA_marshal.trajectory[stepStart + CUDA_kernel.VAR_COUNT]),
            &(CUDA_marshal.parameterVariations[variation * CUDA_kernel.PARAM_COUNT]),
            CUDA_kernel.stepSize);
    }

    // Analysis

    if (M(LLE).toCompute)
    {
        LLE_Settings lle_settings(MS(LLE, 0), MS(LLE, 1), MS(LLE, 2));
        lle_settings.Use3DNorm();
        LLE(data, lle_settings, variation, &finiteDifferenceScheme_hindmarsh_rose, MO(LLE));
    }
}

__device__ void finiteDifferenceScheme_hindmarsh_rose(numb* currentV, numb* nextV, numb* parameters, numb h)
{
    ifMETHOD(P(method), ExplicitEuler)
    {
        Vnext(x) = V(x) + P(stepsize) * (V(y) - P(a) * V(x) * V(x) * V(x) + P(b) * V(x) * V(x) - V(z) + P(Iext));
        Vnext(y) = V(y) + P(stepsize) * (P(c) - P(d) * V(x) * V(x) - V(y));
        Vnext(z) = V(z) + P(stepsize) * (P(r) * (P(s) * (V(x) + P(e)) - V(z)));
    }

    ifMETHOD(P(method), ExplicitMidpoint)
    {
        numb xmp = V(x) + P(stepsize) * 0.5 * (V(y) - P(a) * V(x) * V(x) * V(x) + P(b) * V(x) * V(x) - V(z) + P(Iext));
        numb ymp = V(y) + P(stepsize) * 0.5 * (P(c) - P(d) * V(x) * V(x) - V(y));
        numb zmp = V(z) + P(stepsize) * 0.5 * (P(r) * (P(s) * (V(x) + P(e)) - V(z)));

        Vnext(x) = V(x) + P(stepsize) * (ymp - P(a) * xmp * xmp * xmp + P(b) * xmp * xmp - zmp + P(Iext));
        Vnext(y) = V(y) + P(stepsize) * (P(c) - P(d) * xmp * xmp - ymp);
        Vnext(z) = V(z) + P(stepsize) * (P(r) * (P(s) * (xmp + P(e)) - zmp));
    }

    ifMETHOD(P(method), ExplicitRungeKutta4)
    {
        numb kx1 = V(y) - P(a) * V(x) * V(x) * V(x) + P(b) * V(x) * V(x) - V(z) + P(Iext);
        numb ky1 = P(c) - P(d) * V(x) * V(x) - V(y);
        numb kz1 = P(r) * (P(s) * (V(x) + P(e)) - V(z));

        numb xmp = V(x) + 0.5 * P(stepsize) * kx1;
        numb ymp = V(y) + 0.5 * P(stepsize) * ky1;
        numb zmp = V(z) + 0.5 * P(stepsize) * kz1;

        numb kx2 = ymp - P(a) * xmp * xmp * xmp + P(b) * xmp * xmp - zmp + P(Iext);
        numb ky2 = P(c) - P(d) * xmp * xmp - ymp;
        numb kz2 = P(r) * (P(s) * (xmp + P(e)) - zmp);

        xmp = V(x) + 0.5 * P(stepsize) * kx2;
        ymp = V(y) + 0.5 * P(stepsize) * ky2;
        zmp = V(z) + 0.5 * P(stepsize) * kz2;

        numb kx3 = ymp - P(a) * xmp * xmp * xmp + P(b) * xmp * xmp - zmp + P(Iext);
        numb ky3 = P(c) - P(d) * xmp * xmp - ymp;
        numb kz3 = P(r) * (P(s) * (xmp + P(e)) - zmp);

        xmp = V(x) + P(stepsize) * kx3;
        ymp = V(y) + P(stepsize) * ky3;
        zmp = V(z) + P(stepsize) * kz3;

        numb kx4 = ymp - P(a) * xmp * xmp * xmp + P(b) * xmp * xmp - zmp + P(Iext);
        numb ky4 = P(c) - P(d) * xmp * xmp - ymp;
        numb kz4 = P(r) * (P(s) * (xmp + P(e)) - zmp);

        Vnext(x) = V(x) + P(stepsize) * (kx1 + 2.0 * kx2 + 2.0 * kx3 + kx4) / 6.0;
        Vnext(y) = V(y) + P(stepsize) * (ky1 + 2.0 * ky2 + 2.0 * ky3 + ky4) / 6.0;
        Vnext(z) = V(z) + P(stepsize) * (kz1 + 2.0 * kz2 + 2.0 * kz3 + kz4) / 6.0;
    }

    ifMETHOD(P(method), VariableSymmetryCD)
    {
        numb h1 = 0.5 * P(stepsize) - P(symmetry);
        numb h2 = 0.5 * P(stepsize) + P(symmetry);

        numb xmp = V(x) + h1 * (V(y) - P(a) * V(x) * V(x) * V(x) + P(b) * V(x) * V(x) - V(z) + P(Iext));
        numb ymp = V(y) + h1 * (P(c) - P(d) * xmp * xmp - V(y));
        numb zmp = V(z) + h1 * (P(r) * (P(s) * (xmp + P(e)) - V(z)));

        Vnext(z) = (zmp + P(r) * P(s) * (xmp + P(e)) * h2) / (1 + P(r) * h2);
        Vnext(y) = (ymp + (P(c) - P(d) * xmp * xmp) * h2) / (1 + h2);

        Vnext(x) = xmp + h2 * (Vnext(y) - P(a) * xmp * xmp * xmp + P(b) * xmp * xmp - Vnext(z) + P(Iext));
        Vnext(x) = xmp + h2 * (Vnext(y) - P(a) * Vnext(x) * Vnext(x) * Vnext(x) + P(b) * Vnext(x) * Vnext(x) - Vnext(z) + P(Iext));
    }
}
