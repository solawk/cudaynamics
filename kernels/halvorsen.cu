#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <fstream>

#include "halvorsen.h"
#include <objects.h>
#include <chrono>
#include <wtypes.h>

#define V0(n) varValues[kernel::n]
#define V(n) data[stepStart + kernel::n]
#define P(n) paramValues[kernel::n]
#define NEXT kernel::VAR_COUNT

namespace kernel
{
    const char* name = "Halvorsen";

    const char* VAR_NAMES[]{ "x", "y", "z" };
    float VAR_VALUES[]{ -1.48f, -1.51f, 2.04f };
    bool VAR_RANGING[]{ true, true, true };
    float VAR_STEPS[]{ 2.0f, 2.0f, 2.0f };
    float VAR_MAX[]{ 29.0f, 29.0f, 29.0f };
    int VAR_STEP_COUNTS[]{ 0, 0, 0 };

    const char* PARAM_NAMES[]{ "alpha" };
    float PARAM_VALUES[]{ 1.89f };
    bool PARAM_RANGING[]{ false };
    float PARAM_STEPS[]{ 1.0f };
    float PARAM_MAX[]{ 19.0f };
    int PARAM_STEP_COUNTS[]{ 0 };

    bool executeOnLaunch = true;
    int steps = 1000;
    float stepSize = 0.01f;
    bool onlyShowLast = false;
}

const int THREADS_PER_BLOCK = 256;

__global__ void kernelProgram(float* data, float* params, PreRanging* ranging, int steps, float h, int variationSize)
{
    int b = blockIdx.x;                                     // Current block of THREADS_PER_BLOCK threads
    int t = threadIdx.x;                                    // Current thread in the block, from 0 to THREADS_PER_BLOCK-1
    int variation = (b * THREADS_PER_BLOCK) + t;            // Variation (parameter combination) index
    if (variation >= ranging->totalVariations) return;      // Shutdown thread if there isn't a variation to compute
    int variationStart = variation * variationSize;         // Start index to store the modelling data for the variation
    int stepStart = variationStart;                         // Start index for the current modelling step

    float varValues[kernel::VAR_COUNT];
    float paramValues[kernel::PARAM_COUNT];
    for (int i = 0; i < kernel::VAR_COUNT; i++) varValues[i] = data[i];
    for (int i = 0; i < kernel::PARAM_COUNT; i++) paramValues[i] = params[i];

    // Editing initial state and parameters from ranging
    int tVariation = variation;
    for (int i = ranging->rangingCount - 1; i >= 0; i--)
    {
        bool isVar = ranging->rangings[i].index < kernel::VAR_COUNT;
        int csteps = ranging->rangings[i].steps;
        int step = tVariation % csteps;
        tVariation = tVariation / csteps;
        float value = ranging->rangings[i].min + ranging->rangings[i].step * step;
        
        if (isVar)
        {
            varValues[ranging->rangings[i].index] = value;
        }
        else
        {
            paramValues[ranging->rangings[i].index - kernel::VAR_COUNT] = value;
        }
    }

    // Copying initial state to other variations
    V(x) = V0(x);
    V(y) = V0(y);
    V(z) = V0(z);

    for (int i = 0; i < steps; i++)
    {
        stepStart = variationStart + i * NEXT;

        float dx = -P(alpha)*V(x) - 4*V(y) - 4*V(z) - V(y)*V(y);
        float dy = -P(alpha)*V(y) - 4*V(z) - 4*V(x) - V(z)*V(z);
        float dz = -P(alpha)*V(z) - 4*V(x) - 4*V(y) - V(x)*V(x);

        V(x + NEXT) = V(x) + h * dx;
        V(y + NEXT) = V(y) + h * dy;
        V(z + NEXT) = V(z) + h * dz;
    }

    /*data[variationStart + 0] = b;
    data[variationStart + 1] = t;
    data[variationStart + 2] = variation;*/
}

hipError_t execute(float* data, int rangingCount, int variationSize, int variations, unsigned long int size)
{
    std::chrono::steady_clock::time_point before = std::chrono::steady_clock::now();
    std::chrono::steady_clock::time_point precompute, incompute, postcompute;

    hipError_t cudaStatus;

    int blocks = (int)ceil((float)variations / THREADS_PER_BLOCK);
    int threads = THREADS_PER_BLOCK;

    PreRanging ranging(kernel::VAR_COUNT, kernel::PARAM_COUNT, rangingCount, variations);
    int rangingIndex = 0;

    float* cuda_data = 0;
    float* cuda_params = 0;
    PreRanging* cuda_ranging = 0;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess){fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");goto Error;}

    // Allocating the data array
    cudaStatus = hipMalloc((void**)&cuda_data, size * sizeof(float));
    if (cudaStatus != hipSuccess){fprintf(stderr, "hipMalloc data failed!");goto Error;}

    // Allocating the parameter array
    cudaStatus = hipMalloc((void**)&cuda_params, kernel::PARAM_COUNT * sizeof(float));
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc params failed!"); goto Error; }

    // Parameters array structure (PreRanging):
    for (int i = 0; i < kernel::VAR_COUNT; i++)
        if (kernel::VAR_RANGING[i])
            ranging.rangings[rangingIndex++].init(i, kernel::VAR_VALUES[i], kernel::VAR_STEPS[i], kernel::VAR_MAX[i], kernel::VAR_STEP_COUNTS[i]);
    for (int i = 0; i < kernel::PARAM_COUNT; i++)
        if (kernel::PARAM_RANGING[i])
            ranging.rangings[rangingIndex++].init(i + kernel::VAR_COUNT, kernel::PARAM_VALUES[i], kernel::PARAM_STEPS[i], kernel::PARAM_MAX[i], kernel::PARAM_STEP_COUNTS[i]);

    // Allocating the ranging struct
    cudaStatus = hipMalloc((void**)&cuda_ranging, sizeof(ranging));
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc ranging failed!"); goto Error; }

    // Copying the initial variable values
    cudaStatus = hipMemcpy(cuda_data, kernel::VAR_VALUES, kernel::VAR_COUNT * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess){fprintf(stderr, "hipMemcpy data failed!");goto Error;}

    // Copying the parameter values
    cudaStatus = hipMemcpy(cuda_params, kernel::PARAM_VALUES, kernel::PARAM_COUNT * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess){fprintf(stderr, "hipMemcpy params failed!");goto Error;}

    // Copying the ranging struct
    cudaStatus = hipMemcpy(cuda_ranging, &ranging, sizeof(ranging), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy ranging failed!"); goto Error; }

    // Kernel execution
    precompute = std::chrono::steady_clock::now();
    kernelProgram <<< blocks, threads >>> (cuda_data, cuda_params, cuda_ranging, kernel::steps, kernel::stepSize, variationSize);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess){fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));goto Error;}

    // Awaiting kernel execution end
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess){fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);goto Error;}
    incompute = std::chrono::steady_clock::now();

    // Copying the computed data back
    cudaStatus = hipMemcpy(data, cuda_data, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess){fprintf(stderr, "hipMemcpy failed!");goto Error;}

Error:
    hipFree(cuda_data);

    postcompute = std::chrono::steady_clock::now();
    printf("Precompute time: %i ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(precompute - before).count());
    printf("Incompute time: %i ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(incompute - precompute).count());
    printf("Postcompute time: %i ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(postcompute - incompute).count());

    return cudaStatus;
}

int compute(void** dest, PostRanging* rangingData, HANDLE* writeSemaphore)
{
    std::chrono::steady_clock::time_point before = std::chrono::steady_clock::now();
    std::chrono::steady_clock::time_point after;

    // Preparation

    unsigned long int variations = 1; // Parameter/variable variations (ranging steps)
    int currentVariations; // Variations of the current parameter/variable
    int rangingCount = 0;
    rangingData->clear();

    for (int i = 0; i < kernel::VAR_COUNT; i++)
    {
        if (kernel::VAR_RANGING[i])
        {
            currentVariations = (int)((kernel::VAR_MAX[i] - kernel::VAR_VALUES[i]) / kernel::VAR_STEPS[i]) + 1;
            kernel::VAR_STEP_COUNTS[i] = currentVariations;
            variations *= currentVariations;

            rangingData->names.push_back(kernel::VAR_NAMES[i]);
            rangingData->min.push_back(kernel::VAR_VALUES[i]);
            rangingData->step.push_back(kernel::VAR_STEPS[i]);
            rangingData->max.push_back(kernel::VAR_MAX[i]);
            rangingData->stepCount.push_back(currentVariations);
            rangingData->currentStep.push_back(0);
            rangingData->currentValue.push_back(0);
            rangingCount++;
        }
        else
            kernel::VAR_STEP_COUNTS[i] = 0;
    }

    for (int i = 0; i < kernel::PARAM_COUNT; i++)
    {
        if (kernel::PARAM_RANGING[i])
        {
            currentVariations = (int)((kernel::PARAM_MAX[i] - kernel::PARAM_VALUES[i]) / kernel::PARAM_STEPS[i]) + 1;
            kernel::PARAM_STEP_COUNTS[i] = currentVariations;
            variations *= currentVariations;

            rangingData->names.push_back(kernel::PARAM_NAMES[i]);
            rangingData->min.push_back(kernel::PARAM_VALUES[i]);
            rangingData->step.push_back(kernel::PARAM_STEPS[i]);
            rangingData->max.push_back(kernel::PARAM_MAX[i]);
            rangingData->stepCount.push_back(currentVariations);
            rangingData->currentStep.push_back(0);
            rangingData->currentValue.push_back(0);
            rangingCount++;
        }
        else
            kernel::PARAM_STEP_COUNTS[i] = 0;
    }
    
    rangingData->rangingCount = rangingCount;
    rangingData->totalVariations = variations;
    unsigned long int variationSize = kernel::VAR_COUNT * (kernel::steps + 1); // All steps for the current parameter/variable value combination
    unsigned long int size = variationSize * variations; // Entire data array size
    float* data = new float[size];
    bool hasFailed = false;

    // Execution

    hipError_t cudaStatus = execute(data, rangingCount, variationSize, variations, size);
    if (cudaStatus != hipSuccess){fprintf(stderr, "execute failed!\n");hasFailed = true;}

    // Output

    after = std::chrono::steady_clock::now();
    std::chrono::steady_clock::duration elapsed = after - before;
    printf("CUDA ended in %i ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(elapsed).count());

    /*for (int i = 0; i < kernel::steps + 1; i++)
    {
        printf("%f %f %f\n", data[i * NEXT + kernel::x], data[i * NEXT + kernel::y], data[i * NEXT + kernel::z]);
    }*/

#define WRITE 0
#if WRITE
    std::ofstream outputFile;
    outputFile.open("lorentz.txt", std::ios::out);
    for (unsigned long int i = 0; i < size / 3; i++)
    {
        outputFile << data[3 * i + kernel::x] << " " << data[3 * i + kernel::y] << " " << data[3 * i + kernel::z] << std::endl;
    }
#endif

    // Resetting

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess){fprintf(stderr, "hipDeviceReset failed!\n");hasFailed = true;}

    *dest = data;
    return hasFailed;
}
