#include "hip/hip_runtime.h"
#include "dbscan.h"


__device__  void Period(Computation* data, DBscan_Settings settings, int variation,  int offset) {
    int variationStart = variation * CUDA_marshal.variationSize;
    numb eps = settings.eps;
    int analysedVariable = settings.analysedVariable;
    numb coefPeaks = settings.CoefPeaks;
    numb coefIntervals = settings.CoefIntervals;

    int varCount = CUDA_kernel.VAR_COUNT;
    int variationSize = CUDA_marshal.variationSize;

    // Buffer to hold peak data (amplitudes and indices)
    constexpr int MAX_PEAKS = 512;
    numb peakAmplitudes[MAX_PEAKS];
    numb peakIntervals[MAX_PEAKS];

    int peakCount = 0;
    numb tempPeakAmp = 0, tempPeakTime = 0; bool tempPeakFound = false;
    bool firstpeakreached = false;
    numb temppeakindex;
    numb* computedVariation = CUDA_marshal.trajectory + variationStart;
    for (int i = 1; i < variationSize / varCount - 1 && peakCount < MAX_PEAKS; i++)
    {
        numb prev = computedVariation[analysedVariable + varCount * i - varCount];
        numb curr = computedVariation[analysedVariable + varCount * i];
        numb next = computedVariation[analysedVariable + varCount * i + varCount];
        if (curr > prev && curr > next)
        {
            tempPeakFound = false;
            if (firstpeakreached == false)
            {
                firstpeakreached = true;
                temppeakindex = (float)i;
            }
            else
            {

                peakAmplitudes[peakCount] = curr;
                peakIntervals[peakCount] = (i - temppeakindex) * CUDA_kernel.stepSize;
                peakCount++;
                temppeakindex = (float)i;
            }
        }
        else if (curr == next && curr > prev) {
            tempPeakFound = true; tempPeakAmp = curr; tempPeakTime = i;
        }
        else if (curr < next) {
            tempPeakFound = false;
        }
        else if (curr > next && tempPeakFound) {
            if (firstpeakreached) {
                peakAmplitudes[peakCount] = tempPeakAmp;  peakIntervals[peakCount] = (tempPeakTime - temppeakindex) * CUDA_kernel.stepSize;
                peakCount++;
                temppeakindex = (float)tempPeakTime;
                tempPeakFound = false;
            }
            else {
                firstpeakreached = true;
                temppeakindex = (float)tempPeakTime;
                tempPeakFound = false;
            }
        }
    }
   
    for (int i = 0; i < peakCount-1; i++) {
        peakIntervals[i] *= coefIntervals; peakAmplitudes[i] *= coefPeaks;
    }

    int cluster = 0;
    int NumNeibor = 0;
    int helpfulArray[MAX_PEAKS];
    for (int i = 0; i < MAX_PEAKS; ++i) {
        helpfulArray[i] = 0;
    }

     for (int i = 0; i < peakCount; i++)
        if (NumNeibor >= 1)
        {
            i = helpfulArray[peakCount + NumNeibor - 1];
            helpfulArray[peakCount + NumNeibor - 1] = 0;
            NumNeibor = NumNeibor - 1;
            for (int k = 0; k < peakCount - 1; k++) {
                if (i != k && helpfulArray[k] == 0) {
                    if (sqrt(pow(peakAmplitudes[i] - peakAmplitudes[k], 2) + pow(peakIntervals[i] - peakIntervals[k], 2)) <= eps) {
                        helpfulArray[k] = cluster;
                        helpfulArray[peakCount + k] = k;
                        ++NumNeibor;
                    }
                }

            }
        }
        else if (helpfulArray[i] == 0) {
            NumNeibor = 0;
            ++cluster;
            helpfulArray[i] = cluster;
            for (int k = 0; k < peakCount - 1; k++) {
                if (i != k && helpfulArray[peakCount + k] == 0) {
                    if (sqrt(pow(peakAmplitudes[i] - peakAmplitudes[k], 2) + pow(peakIntervals[i] - peakIntervals[k], 2)) <= eps) {
                        helpfulArray[k] = cluster;
                        helpfulArray[peakCount + k] = k;
                        ++NumNeibor;
                    }
                }

            }
        }  
     cluster--;

     numb mapValue = cluster;

     if (CUDA_kernel.mapWeight == 0.0f)
     {
         numb existingValue = CUDA_marshal.maps[mapPosition] * data->bufferNo;
         CUDA_marshal.maps[mapPosition] = (existingValue + mapValue) / (data->bufferNo + 1);
     }
     else if (CUDA_kernel.mapWeight == 1.0f)
     {
         CUDA_marshal.maps[mapPosition] = mapValue;
     }
     else
     {
         CUDA_marshal.maps[mapPosition] = CUDA_marshal.maps[mapPosition] * (1.0f - CUDA_kernel.mapWeight) + mapValue * CUDA_kernel.mapWeight;
     }
    
}

int DBSCAN::run()
{
    int clusterID = 1;
    vector<Point>::iterator iter;
    for(iter = m_points.begin(); iter != m_points.end(); ++iter)
    {
        if ( iter->clusterID == UNCLASSIFIED )
        {
            if ( expandCluster(*iter, clusterID) != FAILURE )
            {
                clusterID += 1; clusterCount++;
            }
        }
    }
    for (iter = m_points.begin(); iter != m_points.end(); ++iter) {
        if (iter->clusterID == NOISE || iter->clusterID == FAILURE || iter->clusterID == UNCLASSIFIED) {
            clusterCount++;
        }
    }

    return 0;
}

int DBSCAN::expandCluster(Point point, int clusterID)
{    
    vector<int> clusterSeeds = calculateCluster(point);

    if ( clusterSeeds.size() < m_minPoints )
    {
        point.clusterID = NOISE;
        return FAILURE;
    }
    else
    {
        int index = 0, indexCorePoint = 0;
        vector<int>::iterator iterSeeds;
        for( iterSeeds = clusterSeeds.begin(); iterSeeds != clusterSeeds.end(); ++iterSeeds)
        {
            m_points.at(*iterSeeds).clusterID = clusterID;
            if (m_points.at(*iterSeeds).x == point.x && m_points.at(*iterSeeds).y == point.y )
            {
                indexCorePoint = index;
            }
            ++index;
        }
        clusterSeeds.erase(clusterSeeds.begin()+indexCorePoint);

        for( vector<int>::size_type i = 0, n = clusterSeeds.size(); i < n; ++i )
        {
            vector<int> clusterNeighors = calculateCluster(m_points.at(clusterSeeds[i]));

            if ( clusterNeighors.size() >= m_minPoints )
            {
                vector<int>::iterator iterNeighors;
                for ( iterNeighors = clusterNeighors.begin(); iterNeighors != clusterNeighors.end(); ++iterNeighors )
                {
                    if ( m_points.at(*iterNeighors).clusterID == UNCLASSIFIED || m_points.at(*iterNeighors).clusterID == NOISE )
                    {
                        if ( m_points.at(*iterNeighors).clusterID == UNCLASSIFIED )
                        {
                            clusterSeeds.push_back(*iterNeighors);
                            n = clusterSeeds.size();
                        }
                        m_points.at(*iterNeighors).clusterID = clusterID;
                    }
                }
            }
        }

        return SUCCESS;
    }
}

vector<int> DBSCAN::calculateCluster(Point point)
{
    int index = 0;
    vector<Point>::iterator iter;
    vector<int> clusterIndex;
    for( iter = m_points.begin(); iter != m_points.end(); ++iter)
    {
        if ( calculateDistance(point, *iter) <= m_epsilon )
        {
            clusterIndex.push_back(index);
        }
        index++;
    }
    return clusterIndex;
}

inline double DBSCAN::calculateDistance(const Point& pointCore, const Point& pointTarget )
{
    return pow(pointCore.x - pointTarget.x,2)+pow(pointCore.y - pointTarget.y,2);
}



